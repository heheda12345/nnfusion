#include "hip/hip_runtime.h"
#include <stdexcept>
#include <assert.h>
#include <hipblas.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include <sstream>
#include "nnfusion_rt.h"
#include <fstream>

__global__ void printTensor(float* data, int n) {
    for (int i = 0; i < min(n, 10); i++) printf("%f ", data[i]);
    if (n > 10) printf("... %f", data[n-1]);
}

#define DEBUG_TENSOR(tensor, size) { \
    printf("%s: ", #tensor); \
    printTensor<<<1, 1>>>(tensor, size); \
    CUDA_SAFE_CALL(hipDeviceSynchronize()); \
    fflush(stdout); \
    printf("\n"); \
}

__global__ void printTensorChar(char* data, int n) {
    for (int i = 0; i < min(n, 10); i++) printf("%d ", (int) data[i]);
    if (n > 10) printf("... %d", (int) data[n-1]);
}

#define DEBUG_TENSOR_CHAR(tensor, size) { \
    printf("%s: ", #tensor); \
    printTensorChar<<<1, 1>>>(tensor, size); \
    CUDA_SAFE_CALL(hipDeviceSynchronize()); \
    fflush(stdout); \
    printf("\n"); \
}

__global__ void printTensor3D(float* data, int n, int c, int stride) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < c; j++) {
            printf("%f ", data[(j * n + i) * stride]);
        }
        printf("\n");
    }
    // if (n > 10) printf("... %f", data[n-1]);
}

#define DEBUG_TENSOR1(tensor, n, c, stride) { \
    printf("%s: ", #tensor); \
    printTensor3D<<<1, 1>>>(tensor, n, c, stride); \
    CUDA_SAFE_CALL(hipDeviceSynchronize()); \
    fflush(stdout); \
    printf("\n"); \
}
#define MIN(a,b) ((a)>(b)?(b):(a))
#define CUDA_SAFE_CALL(x)                                                                          \
    do                                                                                             \
    {                                                                                              \
        hipError_t result = (x);                                                                  \
        if (result != hipSuccess)                                                                 \
        {                                                                                          \
            const char* msg = hipGetErrorString(result);                                          \
            std::stringstream safe_call_ss;                                                        \
            safe_call_ss << "\nerror: " #x " failed with error"                                    \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg;  \
            throw std::runtime_error(safe_call_ss.str());                                          \
        }                                                                                          \
    } while (0)
#ifndef __HALF_COMPARE_EX__
#define __HALF_COMPARE_EX__
inline __device__ half max(half x, half y) { return x > y ? x : y; }
inline __device__ half min(half x, half y) { return x < y ? x : y; }
#endif
#define CUBLAS_SAFE_CALL(func)                                                                     \
    do                                                                                             \
    {                                                                                              \
        hipblasStatus_t e = (func);                                                                 \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                            \
        {                                                                                          \
            std::stringstream safe_call_ss;                                                        \
            safe_call_ss << "\nerror: " #func " failed with error"                                 \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e;    \
            throw std::runtime_error(safe_call_ss.str());                                          \
        }                                                                                          \
    } while (0)
   #define CUDNN_SAFE_CALL(func)                                                                      \
    do                                                                                             \
    {                                                                                              \
        hipdnnStatus_t e = (func);                                                                  \
        if (e != HIPDNN_STATUS_SUCCESS)                                                             \
        {                                                                                          \
            const char* msg = hipdnnGetErrorString(e);                                              \
            std::stringstream safe_call_ss;                                                        \
            safe_call_ss << "\nerror: " #func " failed with error"                                 \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg;  \
            throw std::runtime_error(safe_call_ss.str());                                          \
        }                                                                                          \
    } while (0)
__device__ __forceinline__ char logical_and(char x0, char x1)
{
    return x0 & x1;
}
__device__ __forceinline__ char greater(int64_t x0, int64_t x1)
{
    return x0 > x1;
}

#if CUDA_VERSION < 9000
#define CREATE_SHFL_MASK(mask, predicate) mask = 0u;
#else
#define FULL_WARP_MASK 0xFFFFFFFF
#define CREATE_SHFL_MASK(mask, predicate) \
  mask = __ballot_sync(FULL_WARP_MASK, (predicate))
#endif

__forceinline__ __device__ float CudaShuffleDownSync(unsigned mask, float val,
                                                     int delta,
                                                     int width = 32) {
#if CUDA_VERSION < 9000
  return __shfl_down(val, delta, width);
#else
  return __shfl_down_sync(mask, val, delta, width);
#endif
}

__device__ static float reduceMax(float val, int tid, int blockSize, float* shm) {
  unsigned mask = 0u;
  CREATE_SHFL_MASK(mask, tid < blockSize);

  val = max(val, CudaShuffleDownSync(mask, val, 16));
  val = max(val, CudaShuffleDownSync(mask, val, 8));
  val = max(val, CudaShuffleDownSync(mask, val, 4));
  val = max(val, CudaShuffleDownSync(mask, val, 2));
  val = max(val, CudaShuffleDownSync(mask, val, 1));

  if (tid < warpSize) shm[tid] = 0.;
  __syncthreads();

  if (tid % warpSize == 0) shm[tid / warpSize] = val;
  __syncthreads();

  CREATE_SHFL_MASK(mask, tid < warpSize);

  if (tid < warpSize) {
    val = shm[tid];

    val = max(val, CudaShuffleDownSync(mask, val, 16));
    val = max(val, CudaShuffleDownSync(mask, val, 8));
    val = max(val, CudaShuffleDownSync(mask, val, 4));
    val = max(val, CudaShuffleDownSync(mask, val, 2));
    val = max(val, CudaShuffleDownSync(mask, val, 1));
  }

  return val;
}

__device__ static float reduceSum(float val, int tid, int blockSize, float* shm) {
  unsigned mask = 0u;
  CREATE_SHFL_MASK(mask, tid < blockSize);

  val += CudaShuffleDownSync(mask, val, 16);
  val += CudaShuffleDownSync(mask, val, 8);
  val += CudaShuffleDownSync(mask, val, 4);
  val += CudaShuffleDownSync(mask, val, 2);
  val += CudaShuffleDownSync(mask, val, 1);

  if (tid < warpSize) shm[tid] = 0.;
  __syncthreads();

  if (tid % warpSize == 0) shm[tid / warpSize] = val;

  __syncthreads();

  CREATE_SHFL_MASK(mask, tid < warpSize);

  if (tid < warpSize) {
    val = shm[tid];

    val += CudaShuffleDownSync(mask, val, 16);
    val += CudaShuffleDownSync(mask, val, 8);
    val += CudaShuffleDownSync(mask, val, 4);
    val += CudaShuffleDownSync(mask, val, 2);
    val += CudaShuffleDownSync(mask, val, 1);
  }

  return val;
}
__device__ __forceinline__ float sigmoid(float x0)
{
    return 1 / (1 + expf(-x0));
}
__device__ __forceinline__ int64_t add(int64_t x0, int64_t x1)
{
    return x0 + x1;
}
__device__ __forceinline__ char nnfusion_less(int64_t x0, int64_t x1)
{
    return x0 < x1;
}
__device__ __forceinline__ float mul(float x0, float x1)
{
    return x0 * x1;
}
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}

typedef signed char int8_t;
typedef signed short int16_t;
typedef signed int int32_t;
typedef signed long int int64_t;
typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
typedef unsigned long int uint64_t;
int num_SMs;
char* group_0_CUDA_GPU0_allocator_memory_pool;
float* GatherV2_19_0;
float* GatherV2_82_0;
float* Reshape_22_0;
float* Reshape_21_0;
float* Reshape_26_0;
float* BatchMatMul_24_0;
float* BatchMatMul_28_0;
float* Reshape_25_0;
float* Reshape_29_0;
float* GatherV2_55_0;
float* GatherV2_47_0;
float* GatherV2_39_0;
float* GatherV2_31_0;
float* GatherV2_59_0;
float* GatherV2_51_0;
float* GatherV2_43_0;
float* GatherV2_35_0;
float* Dot_77_0;
float* Add_83_0;
char* Less_94_0;
int64_t* Reshape_86_0;
int64_t* Sum_90_0;
char* Greater_92_0;
char* group_persist_CUDA_GPU0_allocator_memory_pool;
int64_t* Constant_88_0;
float* Constant_0_0;
int64_t* id_1;
float* Constant_11_0;
int64_t* Constant_54_0;
float* Constant_1_0;
float* Reshape_23_0;
float* Constant_2_0;
float* Reshape_27_0;
int64_t* Constant_46_0;
int64_t* Constant_38_0;
int64_t* Constant_30_0;
int64_t* Constant_58_0;
int64_t* Constant_50_0;
int64_t* Constant_42_0;
int64_t* Constant_34_0;
float* Constant_9_0;
float* Constant_10_0;
float* Constant_5_0;
float* Constant_6_0;
float* Constant_3_0;
float* Constant_4_0;
float* Constant_7_0;
float* Constant_8_0;
float* h_2;
float* c_0;
float* Reshape_76_0;
float* Constant_12_0;
int64_t* tensor_84;
int64_t* Result_101_0;
int64_t* Result_100_0;
int64_t* Constant_93_0;
int64_t* Constant_91_0;
int64_t* tensor_87;
char* cond_0;
char* Result_99_0;
int64_t* Result_98_0;
float* Result_97_0;
float* Result_96_0;
// Node name:	Constant_50
// Description:	Constant
// Input:
// Output:
//	- name: Constant_50_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_50(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_50_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_50_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_5
// Description:	Constant
// Input:
// Output:
//	- name: Constant_5_0	type: float	shape: Shape{256}
void Constant_float_cuda_Constant_5(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_5_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_5_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1024];
    bin_file.read(tmp_mem, 1024);
    hipMemcpyAsync(output0, tmp_mem, 1024, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Less_94
// Description:	Less
// Input:
//	- name: id_1	type: int64_t	shape: Shape{}
//	- name: Constant_93_0	type: int64_t	shape: Shape{}
// Output:
//	- name: Less_94_0	type: char	shape: Shape{}
extern "C" __launch_bounds__(1) __global__ void Less_int64_t_int64_t_char_cuda_Less_94(int64_t* input0, int64_t* input1, char* output0)
{
    output0[threadIdx.x] = nnfusion_less(input0[0], input1[0]);

}
extern void Less_int64_t_int64_t_char_cuda_Less_94_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, int64_t* input0, int64_t* input1, char* output0) {
    Less_int64_t_int64_t_char_cuda_Less_94<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Node name:	Result_97
// Description:	Result
// Input:
//	- name: c_0	type: float	shape: Shape{64, 256}
// Output:
//	- name: Result_97_0	type: float	shape: Shape{64, 256}
void Result_float_float_cuda_lib_Result_97(hipStream_t stream, float* input0, float* output0)
{
    if (input0 != output0)
        CUDA_SAFE_CALL(hipMemcpyAsync(output0, input0,65536, hipMemcpyDeviceToDevice, stream));
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Constant_8_0	type: float	shape: Shape{256}
//	- name: GatherV2_51_0	type: float	shape: Shape{64, 256}
//	- name: Constant_7_0	type: float	shape: Shape{256}
//	- name: GatherV2_47_0	type: float	shape: Shape{64, 256}
//	- name: Constant_4_0	type: float	shape: Shape{256}
//	- name: GatherV2_35_0	type: float	shape: Shape{64, 256}
//	- name: Constant_3_0	type: float	shape: Shape{256}
//	- name: GatherV2_31_0	type: float	shape: Shape{64, 256}
//	- name: Constant_6_0	type: float	shape: Shape{256}
//	- name: GatherV2_43_0	type: float	shape: Shape{64, 256}
//	- name: Constant_5_0	type: float	shape: Shape{256}
//	- name: GatherV2_39_0	type: float	shape: Shape{64, 256}
//	- name: Parameter_14_0	type: float	shape: Shape{64, 256}
//	- name: Constant_10_0	type: float	shape: Shape{256}
//	- name: GatherV2_59_0	type: float	shape: Shape{64, 256}
//	- name: Constant_9_0	type: float	shape: Shape{256}
//	- name: GatherV2_55_0	type: float	shape: Shape{64, 256}
// Output:
//	- name: c_0	type: float	shape: Shape{64, 256}
//	- name: h_2	type: float	shape: Shape{64, 256}
// Fused functions:
// Broadcast, Broadcast_52
// Add, hh2_0
// Broadcast, Broadcast_48
// Add, ih2_0
// Add, Tanh_arg00_0
// Tanh, cellgate_0
// Broadcast, Broadcast_36
// Add, hh0_0
// Broadcast, Broadcast_32
// Add, ih0_0
// Add, Sigmoid_arg00_0
// Sigmoid, ingate_0
// Multiply, @tmp_29
// Broadcast, Broadcast_44
// Add, hh1_0
// Broadcast, Broadcast_40
// Add, ih1_0
// Add, Sigmoid_arg00_1
// Sigmoid, forgetgate_0
// Multiply, @tmp_28
// Add, c_0
// Tanh, @tmp_30
// Broadcast, Broadcast_60
// Add, hh3_0
// Broadcast, Broadcast_56
// Add, ih3_0
// Add, Sigmoid_arg00_2
// Sigmoid, outgate_0
// Multiply, h_2
extern "C" __launch_bounds__(256) __global__ void FusedKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Broadcast_Add_Broadcast_Add_Add_Tanh_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_Add_Tanh_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_0(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* input12, float* input13, float* input14, float* input15, float* input16, float* output0, float* output1)
{
    int tid = blockIdx.x * 256 + threadIdx.x;
    float temp0 = input0[tid % 256];
    float temp1 = add(input1[tid], temp0);
    float temp2 = input2[tid % 256];
    float temp3 = add(input3[tid], temp2);
    float temp4 = add(temp3, temp1);
    float temp5 = tanhf(temp4);
    float temp6 = input4[tid % 256];
    float temp7 = add(input5[tid], temp6);
    float temp8 = input6[tid % 256];
    float temp9 = add(input7[tid], temp8);
    float temp10 = add(temp9, temp7);
    float temp11 = sigmoid(temp10);
    float temp12 = mul(temp11, temp5);
    float temp13 = input8[tid % 256];
    float temp14 = add(input9[tid], temp13);
    float temp15 = input10[tid % 256];
    float temp16 = add(input11[tid], temp15);
    float temp17 = add(temp16, temp14);
    float temp18 = sigmoid(temp17);
    float temp19 = mul(temp18, input12[tid]);
    float temp20 = add(temp19, temp12);
    float temp21 = tanhf(temp20);
    float temp22 = input13[tid % 256];
    float temp23 = add(input14[tid], temp22);
    float temp24 = input15[tid % 256];
    float temp25 = add(input16[tid], temp24);
    float temp26 = add(temp25, temp23);
    float temp27 = sigmoid(temp26);
    float temp28 = mul(temp27, temp21);
    output1[tid] = temp28;
    output0[tid] = temp20;

}
extern void FusedKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Broadcast_Add_Broadcast_Add_Add_Tanh_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_Add_Tanh_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_0_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* input12, float* input13, float* input14, float* input15, float* input16, float* output0, float* output1) {
    FusedKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Broadcast_Add_Broadcast_Add_Add_Tanh_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_Add_Tanh_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_0<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, input11, input12, input13, input14, input15, input16, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Reshape_29_0	type: float	shape: Shape{4, 64, 256}
//	- name: Constant_34_0	type: int64_t	shape: Shape{}
//	- name: Constant_42_0	type: int64_t	shape: Shape{}
//	- name: Constant_50_0	type: int64_t	shape: Shape{}
//	- name: Constant_58_0	type: int64_t	shape: Shape{}
//	- name: Reshape_25_0	type: float	shape: Shape{4, 64, 256}
//	- name: Constant_30_0	type: int64_t	shape: Shape{}
//	- name: Constant_38_0	type: int64_t	shape: Shape{}
//	- name: Constant_46_0	type: int64_t	shape: Shape{}
//	- name: Constant_54_0	type: int64_t	shape: Shape{}
// Output:
//	- name: GatherV2_35_0	type: float	shape: Shape{64, 256}
//	- name: GatherV2_43_0	type: float	shape: Shape{64, 256}
//	- name: GatherV2_51_0	type: float	shape: Shape{64, 256}
//	- name: GatherV2_59_0	type: float	shape: Shape{64, 256}
//	- name: GatherV2_31_0	type: float	shape: Shape{64, 256}
//	- name: GatherV2_39_0	type: float	shape: Shape{64, 256}
//	- name: GatherV2_47_0	type: float	shape: Shape{64, 256}
//	- name: GatherV2_55_0	type: float	shape: Shape{64, 256}
// Fused functions:
// GatherV2_float_int64_t_float_cuda_GatherV2_35<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_29_0, Constant_34_0, GatherV2_35_0);
// GatherV2_float_int64_t_float_cuda_GatherV2_43<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_29_0, Constant_42_0, GatherV2_43_0);
// GatherV2_float_int64_t_float_cuda_GatherV2_51<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_29_0, Constant_50_0, GatherV2_51_0);
// GatherV2_float_int64_t_float_cuda_GatherV2_59<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_29_0, Constant_58_0, GatherV2_59_0);
// GatherV2_float_int64_t_float_cuda_GatherV2_31<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_25_0, Constant_30_0, GatherV2_31_0);
// GatherV2_float_int64_t_float_cuda_GatherV2_39<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_25_0, Constant_38_0, GatherV2_39_0);
// GatherV2_float_int64_t_float_cuda_GatherV2_47<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_25_0, Constant_46_0, GatherV2_47_0);
// GatherV2_float_int64_t_float_cuda_GatherV2_55<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_25_0, Constant_54_0, GatherV2_55_0);
// Deduped function map: <src_function_name : deduped_function_name>
// GatherV2_float_int64_t_float_cuda_GatherV2_43 : GatherV2_float_int64_t_float_cuda_GatherV2_35
// GatherV2_float_int64_t_float_cuda_GatherV2_51 : GatherV2_float_int64_t_float_cuda_GatherV2_35
// GatherV2_float_int64_t_float_cuda_GatherV2_59 : GatherV2_float_int64_t_float_cuda_GatherV2_35
// GatherV2_float_int64_t_float_cuda_GatherV2_31 : GatherV2_float_int64_t_float_cuda_GatherV2_35
// GatherV2_float_int64_t_float_cuda_GatherV2_39 : GatherV2_float_int64_t_float_cuda_GatherV2_35
// GatherV2_float_int64_t_float_cuda_GatherV2_47 : GatherV2_float_int64_t_float_cuda_GatherV2_35
// GatherV2_float_int64_t_float_cuda_GatherV2_55 : GatherV2_float_int64_t_float_cuda_GatherV2_35

// Node name:	GatherV2_35
// Description:	GatherV2
// Input:
//	- name: Reshape_29_0	type: float	shape: Shape{4, 64, 256}
//	- name: Constant_34_0	type: int64_t	shape: Shape{}
// Output:
//	- name: GatherV2_35_0	type: float	shape: Shape{64, 256}
__device__ __noinline__ void GatherV2_float_int64_t_float_cuda_GatherV2_35_block_kernel(float* input0, int64_t* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(256, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    float* params = input0;
    int64_t* indices = input1;
    float* out = output0;
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 16384)
    {
        uint32_t batch_i = 0;
        uint32_t indices_i = 0;
        uint32_t slice_i = 0;
        indices_i = i / 16384;
        slice_i = i - indices_i * 16384;
        uint32_t gather_i = *(indices + indices_i);
        if (gather_i >= 4)
           out[i] = 0;
        else
        {
            uint32_t params_i = (batch_i * 4 + gather_i) * 16384 + slice_i;
            out[i] = __ldg(params + params_i);
        }
    }

}
extern "C" __global__  void BlockFusionKernel_float_int64_t_int64_t_int64_t_int64_t_float_int64_t_int64_t_int64_t_int64_t_float_float_float_float_float_float_float_float_cuda_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_2(float* input0, int64_t* input1, int64_t* input2, int64_t* input3, int64_t* input4, float* input5, int64_t* input6, int64_t* input7, int64_t* input8, int64_t* input9, float* output0, float* output1, float* output2, float* output3, float* output4, float* output5, float* output6, float* output7)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_35_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_35_block_kernel(input0, input2, output1, threadIdx.x, blockIdx.x - 64 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_35_block_kernel(input0, input3, output2, threadIdx.x, blockIdx.x - 128 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_35_block_kernel(input0, input4, output3, threadIdx.x, blockIdx.x - 192 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 319)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_35_block_kernel(input5, input6, output4, threadIdx.x, blockIdx.x - 256 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 320 && (int)blockIdx.x <= 383)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_35_block_kernel(input5, input7, output5, threadIdx.x, blockIdx.x - 320 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 384 && (int)blockIdx.x <= 447)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_35_block_kernel(input5, input8, output6, threadIdx.x, blockIdx.x - 384 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 448 && (int)blockIdx.x <= 511)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_35_block_kernel(input5, input9, output7, threadIdx.x, blockIdx.x - 448 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_int64_t_int64_t_int64_t_int64_t_float_int64_t_int64_t_int64_t_int64_t_float_float_float_float_float_float_float_float_cuda_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_2_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, int64_t* input1, int64_t* input2, int64_t* input3, int64_t* input4, float* input5, int64_t* input6, int64_t* input7, int64_t* input8, int64_t* input9, float* output0, float* output1, float* output2, float* output3, float* output4, float* output5, float* output6, float* output7) {
    BlockFusionKernel_float_int64_t_int64_t_int64_t_int64_t_float_int64_t_int64_t_int64_t_int64_t_float_float_float_float_float_float_float_float_cuda_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_2<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, output0, output1, output2, output3, output4, output5, output6, output7);
}
// Node name:	And_95
// Description:	And
// Input:
//	- name: Greater_92_0	type: char	shape: Shape{}
//	- name: Less_94_0	type: char	shape: Shape{}
// Output:
//	- name: cond_0	type: char	shape: Shape{}
extern "C" __launch_bounds__(1) __global__ void And_char_char_char_cuda_And_95(char* input0, char* input1, char* output0)
{
    output0[threadIdx.x] = logical_and(input0[0], input1[0]);

}
extern void And_char_char_char_cuda_And_95_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, char* input0, char* input1, char* output0) {
    And_char_char_char_cuda_And_95<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Node name:	Result_101
// Description:	Result
// Input:
//	- name: tensor_84	type: int64_t	shape: Shape{64}
// Output:
//	- name: Result_101_0	type: int64_t	shape: Shape{64}
void Result_int64_t_int64_t_cuda_lib_Result_101(hipStream_t stream, int64_t* input0, int64_t* output0)
{
    if (input0 != output0)
        CUDA_SAFE_CALL(hipMemcpyAsync(output0, input0,512, hipMemcpyDeviceToDevice, stream));
}
// Node name:	 BlockFusion
// Input:
//	- name: Reshape_26_0	type: float	shape: Shape{64, 256}
//	- name: Reshape_27_0	type: float	shape: Shape{4, 256, 256}
//	- name: Reshape_22_0	type: float	shape: Shape{64, 256}
//	- name: Reshape_23_0	type: float	shape: Shape{4, 256, 256}
// Output:
//	- name: BatchMatMul_28_0	type: float	shape: Shape{4, 64, 256}
//	- name: BatchMatMul_24_0	type: float	shape: Shape{4, 64, 256}
// Fused functions:
// BatchMatMul_float_float_float_cuda_BatchMatMul_28<<<dim3(8, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_26_0, Reshape_27_0, BatchMatMul_28_0);
// BatchMatMul_float_float_float_cuda_BatchMatMul_24<<<dim3(8, 1, 1), dim3(256, 1, 1), 0, 0>>>(Reshape_22_0, Reshape_23_0, BatchMatMul_24_0);
// Deduped function map: <src_function_name : deduped_function_name>
// BatchMatMul_float_float_float_cuda_BatchMatMul_24 : BatchMatMul_float_float_float_cuda_BatchMatMul_28

// Node name:	BatchMatMul_28
// Description:	BatchMatMul
// Input:
//	- name: Reshape_26_0	type: float	shape: Shape{64, 256}
//	- name: Reshape_27_0	type: float	shape: Shape{4, 256, 256}
// Output:
//	- name: BatchMatMul_28_0	type: float	shape: Shape{4, 64, 256}
__device__ __noinline__ void BatchMatMul_float_float_float_cuda_BatchMatMul_28_block_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ compute, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        for (int i = 0; i < 4; i++) __syncthreads();
        return;
    }
    const dim3 blockDim(256, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    float* A_shared = (float*)(shared_buffer + 0);
    float* B_shared = (float*)(shared_buffer + 16384);
    {
        {
            float compute_local[4];
            compute_local[(0)] = 0.000000e+00f;
            compute_local[(2)] = 0.000000e+00f;
            compute_local[(1)] = 0.000000e+00f;
            compute_local[(3)] = 0.000000e+00f;
            for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
              __syncthreads();
              ((float4*)(A_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(A + (((((((((int)blockIdx.x) & 31) >> 4) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (k_outer_outer * 128)) + ((((int)threadIdx.x) & 31) * 4)))))[0];
              ((float4*)(A_shared + (((((int)threadIdx.x) * 4) + 1024))))[0] = ((float4*)(A + ((((((((((int)blockIdx.x) & 31) >> 4) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (k_outer_outer * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 2048))))[0];
              ((float4*)(A_shared + (((((int)threadIdx.x) * 4) + 2048))))[0] = ((float4*)(A + ((((((((((int)blockIdx.x) & 31) >> 4) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (k_outer_outer * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 4096))))[0];
              ((float4*)(A_shared + (((((int)threadIdx.x) * 4) + 3072))))[0] = ((float4*)(A + ((((((((((int)blockIdx.x) & 31) >> 4) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (k_outer_outer * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 6144))))[0];
              ((float4*)(B_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(B + (((((((((int)blockIdx.x) >> 5) * 131072) + (k_outer_outer * 32768)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
              ((float4*)(B_shared + (((((int)threadIdx.x) * 4) + 1024))))[0] = ((float4*)(B + ((((((((((int)blockIdx.x) >> 5) * 131072) + (k_outer_outer * 32768)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 16384))))[0];
              ((float4*)(B_shared + (((((int)threadIdx.x) * 4) + 2048))))[0] = ((float4*)(B + ((((((((((int)blockIdx.x) >> 5) * 131072) + (k_outer_outer * 32768)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 65536))))[0];
              ((float4*)(B_shared + (((((((((int)threadIdx.x) * 4) + 3072) >> 11) * 2048) + (((((int)threadIdx.x) >> 2) + 64) * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0] = ((float4*)(B + ((((((((((int)blockIdx.x) >> 5) * 131072) + ((((((int)threadIdx.x) * 4) + 3072) >> 11) * 65536)) + (k_outer_outer * 32768)) + (((((int)threadIdx.x) >> 2) + 64) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
              __syncthreads();
              for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
                for (int k_inner = 0; k_inner < 64; ++k_inner) {
                  compute_local[(0)] = (compute_local[(0)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (k_outer_inner * 64)) + k_inner))] * B_shared[((((k_outer_inner * 1024) + (k_inner * 16)) + (((int)threadIdx.x) & 15)))]));
                  compute_local[(2)] = (compute_local[(2)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (k_outer_inner * 64)) + k_inner))] * B_shared[(((((k_outer_inner * 1024) + (k_inner * 16)) + (((int)threadIdx.x) & 15)) + 2048))]));
                  compute_local[(1)] = (compute_local[(1)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 256) + (k_outer_inner * 64)) + k_inner) + 128))] * B_shared[((((k_outer_inner * 1024) + (k_inner * 16)) + (((int)threadIdx.x) & 15)))]));
                  compute_local[(3)] = (compute_local[(3)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 256) + (k_outer_inner * 64)) + k_inner) + 128))] * B_shared[(((((k_outer_inner * 1024) + (k_inner * 16)) + (((int)threadIdx.x) & 15)) + 2048))]));
                }
              }
            }
            for (int x_inner = 0; x_inner < 2; ++x_inner) {
              compute[((((((((((int)blockIdx.x) >> 5) * 32768) + (((((int)blockIdx.x) & 31) >> 4) * 8192)) + ((((int)threadIdx.x) >> 4) * 512)) + (x_inner * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)))] = compute_local[(x_inner)];
              compute[(((((((((((int)blockIdx.x) >> 5) * 32768) + (((((int)blockIdx.x) & 31) >> 4) * 8192)) + ((((int)threadIdx.x) >> 4) * 512)) + (x_inner * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 16384))] = compute_local[((x_inner + 2))];
            }
        }
    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_cuda_BatchMatMul_BatchMatMul_1(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1)
{
    __shared__ char shared_buffer[32768];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        BatchMatMul_float_float_float_cuda_BatchMatMul_28_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        BatchMatMul_float_float_float_cuda_BatchMatMul_28_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_cuda_BatchMatMul_BatchMatMul_1_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_cuda_BatchMatMul_BatchMatMul_1<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1);
}
// Node name:	Reshape_21
// Description:	Reshape
// Input:
//	- name: Parameter_13_0	type: float	shape: Shape{64, 256}
// Output:
//	- name: Reshape_21_0	type: float	shape: Shape{64, 256}
extern "C" __launch_bounds__(256) __global__ void Reshape_float_float_cuda_Reshape_21(float* input0, float* output0)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 16384) { return; }
    output0[tid] = input0[tid];

}
extern void Reshape_float_float_cuda_Reshape_21_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* output0) {
    Reshape_float_float_cuda_Reshape_21<<<grids, blocks, mem, stream>>>(input0, output0);
}
// Node name:	Result_99
// Description:	Result
// Input:
//	- name: cond_0	type: char	shape: Shape{}
// Output:
//	- name: Result_99_0	type: char	shape: Shape{}
void Result_char_char_cuda_lib_Result_99(hipStream_t stream, char* input0, char* output0)
{
    if (input0 != output0)
        CUDA_SAFE_CALL(hipMemcpyAsync(output0, input0,1, hipMemcpyDeviceToDevice, stream));
}
// Node name:	 BlockFusion
// Input:
//	- name: Parameter_16_0	type: int64_t	shape: Shape{50, 64}
//	- name: Reshape_86_0	type: int64_t	shape: Shape{1}
//	- name: tensor_84	type: int64_t	shape: Shape{64}
// Output:
//	- name: tensor_87	type: int64_t	shape: Shape{50, 64}
//	- name: Sum_90_0	type: int64_t	shape: Shape{}
// Fused functions:
// ScatterND_int64_t_int64_t_int64_t_int64_t_cuda_ScatterND_87<<<dim3(1, 1, 1), dim3(256, 1, 1), 0, 0>>>(Parameter_16_0, Reshape_86_0, tensor_84, tensor_87);
// Sum_int64_t_int64_t_cuda_Sum_90<<<dim3(1, 1, 1), dim3(64, 1, 1), 0, 0>>>(tensor_84, Sum_90_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	ScatterND_87
// Description:	ScatterND
// Input:
//	- name: Parameter_16_0	type: int64_t	shape: Shape{50, 64}
//	- name: Reshape_86_0	type: int64_t	shape: Shape{1}
//	- name: tensor_84	type: int64_t	shape: Shape{64}
// Output:
//	- name: tensor_87	type: int64_t	shape: Shape{50, 64}
__device__ __noinline__ void ScatterND_int64_t_int64_t_int64_t_int64_t_cuda_ScatterND_87_block_kernel(int64_t* input0, int64_t* input1, int64_t* input2, int64_t* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(256, 1, 1);
    const dim3 gridDim(1, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 64) { return; }
    input0[64* input1[0] + tid] = input2[tid];

}
// Node name:	Sum_90
// Description:	Sum
// Input:
//	- name: tensor_84	type: int64_t	shape: Shape{64}
// Output:
//	- name: Sum_90_0	type: int64_t	shape: Shape{}
__device__ __noinline__ void Sum_int64_t_int64_t_cuda_Sum_90_block_kernel(int64_t* input0, int64_t* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 64){
        for (int i = 0; i < 2; i++) __syncthreads();
        return;
    }
    const dim3 blockDim(64, 1, 1);
    const dim3 gridDim(1, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

    int width = 64;
    int block_size = 64;
    const int warp_size = 32;
                        float* shm = (float*)(shared_buffer + 128);

    int thread_idx = threadIdx.x;
    int block_idx = blockIdx.x;
    int data_idx_offset = block_idx * width;

    float val = 0.0;
    for (int tidx = thread_idx; tidx < width; tidx += block_size) {
        int data_idx = tidx + data_idx_offset;
        val += input0[data_idx];
    }
    val = reduceSum(val, thread_idx, block_size, shm);
    if (thread_idx == 0) output0[block_idx] = val;


}
extern "C" __global__  void BlockFusionKernel_int64_t_int64_t_int64_t_int64_t_int64_t_cuda_ScatterND_Sum_3(int64_t* input0, int64_t* input1, int64_t* input2, int64_t* output0, int64_t* output1)
{
    __shared__ char shared_buffer[256];

    if (blockIdx.x == 0)
    {
        ScatterND_int64_t_int64_t_int64_t_int64_t_cuda_ScatterND_87_block_kernel(input0, input1, input2, output0, threadIdx.x, 0, shared_buffer);
    }
    else if (blockIdx.x == 1)
    {
        Sum_int64_t_int64_t_cuda_Sum_90_block_kernel(input2, output1, threadIdx.x, 0, shared_buffer);
    }

}
extern void BlockFusionKernel_int64_t_int64_t_int64_t_int64_t_int64_t_cuda_ScatterND_Sum_3_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, int64_t* input0, int64_t* input1, int64_t* input2, int64_t* output0, int64_t* output1) {
    BlockFusionKernel_int64_t_int64_t_int64_t_int64_t_int64_t_cuda_ScatterND_Sum_3<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0, output1);
}
// Node name:	ArgMax_84
// Description:	ArgMax
// Input:
//	- name: Add_83_0	type: float	shape: Shape{64, 3797}
// Output:
//	- name: tensor_84	type: int64_t	shape: Shape{64}
extern "C" __launch_bounds__(256) __global__ void ArgMax_float_int64_t_cuda_ArgMax_84(float* input0, int64_t* output0)
{

    int in_reduce_size = 1;
    int reduce_size = 3797;
    int out_id = blockIdx.x / in_reduce_size;
    int in_id = blockIdx.x % in_reduce_size;
    int bias = out_id * reduce_size * in_reduce_size + in_id;
    int max_id = -1;
    float max_value = -FLT_MAX;
    for (int i = threadIdx.x; i < reduce_size; i += blockDim.x) {
        float value = input0[bias + i * in_reduce_size];
        if (value > max_value) {
            max_value = value;
            max_id = i;
        }
    }
        __shared__ float shared_max_value[256];
    __shared__ int64_t shared_max_id[256];

    shared_max_value[threadIdx.x] = max_value;
    shared_max_id[threadIdx.x] = max_id;
    __syncthreads();
    # pragma unroll
    for (int i = 256 / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            if (shared_max_value[threadIdx.x] < shared_max_value[threadIdx.x + i]) {
                shared_max_value[threadIdx.x] = shared_max_value[threadIdx.x + i];
                shared_max_id[threadIdx.x] = shared_max_id[threadIdx.x + i];
            }
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        output0[out_id * in_reduce_size + in_id] = shared_max_id[0];
    }

}
extern void ArgMax_float_int64_t_cuda_ArgMax_84_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, int64_t* output0) {
    ArgMax_float_int64_t_cuda_ArgMax_84<<<grids, blocks, mem, stream>>>(input0, output0);
}
// Node name:	Reshape_86
// Description:	Reshape
// Input:
//	- name: Parameter_17_0	type: int64_t	shape: Shape{}
// Output:
//	- name: Reshape_86_0	type: int64_t	shape: Shape{1}
extern "C" __launch_bounds__(256) __global__ void Reshape_int64_t_int64_t_cuda_Reshape_86(int64_t* input0, int64_t* output0)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 1) { return; }
    output0[tid] = input0[tid];

}
extern void Reshape_int64_t_int64_t_cuda_Reshape_86_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, int64_t* input0, int64_t* output0) {
    Reshape_int64_t_int64_t_cuda_Reshape_86<<<grids, blocks, mem, stream>>>(input0, output0);
}
// Node name:	 BlockFusion
// Input:
//	- name: Parameter_15_0	type: float	shape: Shape{50, 64, 3797}
//	- name: Parameter_17_0	type: int64_t	shape: Shape{}
//	- name: Constant_0_0	type: float	shape: Shape{3797, 256}
//	- name: Parameter_18_0	type: int64_t	shape: Shape{64}
//	- name: Constant_88_0	type: int64_t	shape: Shape{}
// Output:
//	- name: GatherV2_82_0	type: float	shape: Shape{64, 3797}
//	- name: GatherV2_19_0	type: float	shape: Shape{64, 256}
//	- name: id_1	type: int64_t	shape: Shape{}
// Fused functions:
// GatherV2_float_int64_t_float_cuda_GatherV2_82<<<dim3(950, 1, 1), dim3(256, 1, 1), 0, 0>>>(Parameter_15_0, Parameter_17_0, GatherV2_82_0);
// GatherV2_float_int64_t_float_cuda_GatherV2_19<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Constant_0_0, Parameter_18_0, GatherV2_19_0);
// Add_int64_t_int64_t_int64_t_cuda_Add_89<<<dim3(1, 1, 1), dim3(1, 1, 1), 0, 0>>>(Parameter_17_0, Constant_88_0, id_1);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	GatherV2_82
// Description:	GatherV2
// Input:
//	- name: Parameter_15_0	type: float	shape: Shape{50, 64, 3797}
//	- name: Parameter_17_0	type: int64_t	shape: Shape{}
// Output:
//	- name: GatherV2_82_0	type: float	shape: Shape{64, 3797}
__device__ __noinline__ void GatherV2_float_int64_t_float_cuda_GatherV2_82_block_kernel(float* input0, int64_t* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(256, 1, 1);
    const dim3 gridDim(950, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    float* params = input0;
    int64_t* indices = input1;
    float* out = output0;
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 243008)
    {
        uint32_t batch_i = 0;
        uint32_t indices_i = 0;
        uint32_t slice_i = 0;
        indices_i = i / 243008;
        slice_i = i - indices_i * 243008;
        uint32_t gather_i = *(indices + indices_i);
        if (gather_i >= 50)
           out[i] = 0;
        else
        {
            uint32_t params_i = (batch_i * 50 + gather_i) * 243008 + slice_i;
            out[i] = __ldg(params + params_i);
        }
    }

}
// Node name:	GatherV2_19
// Description:	GatherV2
// Input:
//	- name: Constant_0_0	type: float	shape: Shape{3797, 256}
//	- name: Parameter_18_0	type: int64_t	shape: Shape{64}
// Output:
//	- name: GatherV2_19_0	type: float	shape: Shape{64, 256}
__device__ __noinline__ void GatherV2_float_int64_t_float_cuda_GatherV2_19_block_kernel(float* input0, int64_t* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(256, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    float* params = input0;
    int64_t* indices = input1;
    float* out = output0;
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 16384)
    {
        uint32_t batch_i = 0;
        uint32_t indices_i = 0;
        uint32_t slice_i = 0;
        indices_i = i / 256;
        slice_i = i - indices_i * 256;
        uint32_t gather_i = *(indices + indices_i);
        if (gather_i >= 3797)
           out[i] = 0;
        else
        {
            uint32_t params_i = (batch_i * 3797 + gather_i) * 256 + slice_i;
            out[i] = __ldg(params + params_i);
        }
    }

}
// Node name:	Add_89
// Description:	Add
// Input:
//	- name: Parameter_17_0	type: int64_t	shape: Shape{}
//	- name: Constant_88_0	type: int64_t	shape: Shape{}
// Output:
//	- name: id_1	type: int64_t	shape: Shape{}
__device__ __noinline__ void Add_int64_t_int64_t_int64_t_cuda_Add_89_block_kernel(int64_t* input0, int64_t* input1, int64_t* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 1){
        return;
    }
    const dim3 blockDim(1, 1, 1);
    const dim3 gridDim(1, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[threadIdx.x] = add(input0[0], input1[0]);

}
extern "C" __global__  void BlockFusionKernel_float_int64_t_float_int64_t_int64_t_float_float_int64_t_cuda_GatherV2_GatherV2_Add_0(float* input0, int64_t* input1, float* input2, int64_t* input3, int64_t* input4, float* output0, float* output1, int64_t* output2)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 949)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_82_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 950 && (int)blockIdx.x <= 1013)
    {
        GatherV2_float_int64_t_float_cuda_GatherV2_19_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 950 + 0, NULL);
    }
    else if (blockIdx.x == 1014)
    {
        Add_int64_t_int64_t_int64_t_cuda_Add_89_block_kernel(input1, input4, output2, threadIdx.x, 0, NULL);
    }

}
extern void BlockFusionKernel_float_int64_t_float_int64_t_int64_t_float_float_int64_t_cuda_GatherV2_GatherV2_Add_0_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, int64_t* input1, float* input2, int64_t* input3, int64_t* input4, float* output0, float* output1, int64_t* output2) {
    BlockFusionKernel_float_int64_t_float_int64_t_int64_t_float_float_int64_t_cuda_GatherV2_GatherV2_Add_0<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, output0, output1, output2);
}
// Node name:	Dot_77
// Description:	Dot
// Input:
//	- name: Reshape_76_0	type: float	shape: Shape{64, 256}
//	- name: Constant_11_0	type: float	shape: Shape{3797, 256}
// Output:
//	- name: Dot_77_0	type: float	shape: Shape{64, 3797}
extern "C" __global__ void Dot_float_float_float_cuda_Dot_77(float* __restrict__ A, float* __restrict__ B, float* __restrict__ compute)
{
    __shared__ float A_shared[256];
    __shared__ float B_shared[768];
    {
        {
          float compute_local[8];
          
          
          float A_shared_local[4];
          float B_shared_local[2];
          compute_local[(0)] = 0.000000e+00f;
          compute_local[(2)] = 0.000000e+00f;
          compute_local[(4)] = 0.000000e+00f;
          compute_local[(6)] = 0.000000e+00f;
          compute_local[(1)] = 0.000000e+00f;
          compute_local[(3)] = 0.000000e+00f;
          compute_local[(5)] = 0.000000e+00f;
          compute_local[(7)] = 0.000000e+00f;
          for (int k_outer = 0; k_outer < 32; ++k_outer) {
            __syncthreads();
            if (((int)threadIdx.x) < 256) {
              A_shared[(((int)threadIdx.x))] = A[((((((((int)blockIdx.x) / 40) * 8192) + ((((int)threadIdx.x) >> 3) * 256)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)))];
            }
            B_shared[(((int)threadIdx.x))] = B[((((((((int)blockIdx.x) % 40) * 24576) + ((((int)threadIdx.x) >> 3) * 256)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)))];
            if ((((((int)blockIdx.x) % 40) * 96) + (((int)threadIdx.x) >> 3)) < 3749) {
              B_shared[((((int)threadIdx.x) + 384))] = B[(((((((((int)blockIdx.x) % 40) * 24576) + ((((int)threadIdx.x) >> 3) * 256)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)) + 12288))];
            }
            __syncthreads();
            for (int k_inner_outer = 0; k_inner_outer < 8; ++k_inner_outer) {
              A_shared_local[(0)] = A_shared[((((((int)threadIdx.x) / 48) * 8) + k_inner_outer))];
              A_shared_local[(1)] = A_shared[(((((((int)threadIdx.x) / 48) * 8) + k_inner_outer) + 64))];
              A_shared_local[(2)] = A_shared[(((((((int)threadIdx.x) / 48) * 8) + k_inner_outer) + 128))];
              A_shared_local[(3)] = A_shared[(((((((int)threadIdx.x) / 48) * 8) + k_inner_outer) + 192))];
              B_shared_local[(0)] = B_shared[((((((int)threadIdx.x) % 48) * 8) + k_inner_outer))];
              if ((((((int)blockIdx.x) % 40) * 96) + (((int)threadIdx.x) % 48)) < 3749) {
                B_shared_local[(1)] = B_shared[(((((((int)threadIdx.x) % 48) * 8) + k_inner_outer) + 384))];
              }
              compute_local[(0)] = (compute_local[(0)] + (A_shared_local[(0)] * B_shared_local[(0)]));
              compute_local[(2)] = (compute_local[(2)] + (A_shared_local[(1)] * B_shared_local[(0)]));
              compute_local[(4)] = (compute_local[(4)] + (A_shared_local[(2)] * B_shared_local[(0)]));
              compute_local[(6)] = (compute_local[(6)] + (A_shared_local[(3)] * B_shared_local[(0)]));
              if ((((((int)blockIdx.x) % 40) * 96) + (((int)threadIdx.x) % 48)) < 3749) {
                compute_local[(1)] = (compute_local[(1)] + (A_shared_local[(0)] * B_shared_local[(1)]));
                compute_local[(3)] = (compute_local[(3)] + (A_shared_local[(1)] * B_shared_local[(1)]));
                compute_local[(5)] = (compute_local[(5)] + (A_shared_local[(2)] * B_shared_local[(1)]));
                compute_local[(7)] = (compute_local[(7)] + (A_shared_local[(3)] * B_shared_local[(1)]));
              }
            }
          }
          compute[((((((((int)blockIdx.x) / 40) * 121504) + ((((int)threadIdx.x) / 48) * 3797)) + ((((int)blockIdx.x) % 40) * 96)) + (((int)threadIdx.x) % 48)))] = compute_local[(0)];
          compute[(((((((((int)blockIdx.x) / 40) * 121504) + ((((int)threadIdx.x) / 48) * 3797)) + ((((int)blockIdx.x) % 40) * 96)) + (((int)threadIdx.x) % 48)) + 30376))] = compute_local[(2)];
          compute[(((((((((int)blockIdx.x) / 40) * 121504) + ((((int)threadIdx.x) / 48) * 3797)) + ((((int)blockIdx.x) % 40) * 96)) + (((int)threadIdx.x) % 48)) + 60752))] = compute_local[(4)];
          compute[(((((((((int)blockIdx.x) / 40) * 121504) + ((((int)threadIdx.x) / 48) * 3797)) + ((((int)blockIdx.x) % 40) * 96)) + (((int)threadIdx.x) % 48)) + 91128))] = compute_local[(6)];
          if ((((((int)blockIdx.x) % 40) * 96) + (((int)threadIdx.x) % 48)) < 3749) {
            compute[(((((((((int)blockIdx.x) / 40) * 121504) + ((((int)threadIdx.x) / 48) * 3797)) + ((((int)blockIdx.x) % 40) * 96)) + (((int)threadIdx.x) % 48)) + 48))] = compute_local[(1)];
            compute[(((((((((int)blockIdx.x) / 40) * 121504) + ((((int)threadIdx.x) / 48) * 3797)) + ((((int)blockIdx.x) % 40) * 96)) + (((int)threadIdx.x) % 48)) + 30424))] = compute_local[(3)];
            compute[(((((((((int)blockIdx.x) / 40) * 121504) + ((((int)threadIdx.x) / 48) * 3797)) + ((((int)blockIdx.x) % 40) * 96)) + (((int)threadIdx.x) % 48)) + 60800))] = compute_local[(5)];
            compute[(((((((((int)blockIdx.x) / 40) * 121504) + ((((int)threadIdx.x) / 48) * 3797)) + ((((int)blockIdx.x) % 40) * 96)) + (((int)threadIdx.x) % 48)) + 91176))] = compute_local[(7)];
          }
        }


    }

}
extern void Dot_float_float_float_cuda_Dot_77_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* __restrict__ A, float* __restrict__ B, float* __restrict__ compute) {
    Dot_float_float_float_cuda_Dot_77<<<grids, blocks, mem, stream>>>(A, B, compute);
}
// Node name:	Result_100
// Description:	Result
// Input:
//	- name: id_1	type: int64_t	shape: Shape{}
// Output:
//	- name: Result_100_0	type: int64_t	shape: Shape{}
void Result_int64_t_int64_t_cuda_lib_Result_100(hipStream_t stream, int64_t* input0, int64_t* output0)
{
    if (input0 != output0)
        CUDA_SAFE_CALL(hipMemcpyAsync(output0, input0,8, hipMemcpyDeviceToDevice, stream));
}
// Node name:	Constant_9
// Description:	Constant
// Input:
// Output:
//	- name: Constant_9_0	type: float	shape: Shape{256}
void Constant_float_cuda_Constant_9(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_9_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_9_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1024];
    bin_file.read(tmp_mem, 1024);
    hipMemcpyAsync(output0, tmp_mem, 1024, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3_0	type: float	shape: Shape{256}
void Constant_float_cuda_Constant_3(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1024];
    bin_file.read(tmp_mem, 1024);
    hipMemcpyAsync(output0, tmp_mem, 1024, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_0
// Description:	Constant
// Input:
// Output:
//	- name: Constant_0_0	type: float	shape: Shape{3797, 256}
void Constant_float_cuda_Constant_0(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_0_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_0_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[3888128];
    bin_file.read(tmp_mem, 3888128);
    hipMemcpyAsync(output0, tmp_mem, 3888128, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_12
// Description:	Constant
// Input:
// Output:
//	- name: Constant_12_0	type: float	shape: Shape{3797}
void Constant_float_cuda_Constant_12(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_12_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_12_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[15188];
    bin_file.read(tmp_mem, 15188);
    hipMemcpyAsync(output0, tmp_mem, 15188, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_7
// Description:	Constant
// Input:
// Output:
//	- name: Constant_7_0	type: float	shape: Shape{256}
void Constant_float_cuda_Constant_7(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_7_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_7_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1024];
    bin_file.read(tmp_mem, 1024);
    hipMemcpyAsync(output0, tmp_mem, 1024, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_4
// Description:	Constant
// Input:
// Output:
//	- name: Constant_4_0	type: float	shape: Shape{256}
void Constant_float_cuda_Constant_4(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_4_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_4_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1024];
    bin_file.read(tmp_mem, 1024);
    hipMemcpyAsync(output0, tmp_mem, 1024, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_8
// Description:	Constant
// Input:
// Output:
//	- name: Constant_8_0	type: float	shape: Shape{256}
void Constant_float_cuda_Constant_8(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_8_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_8_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1024];
    bin_file.read(tmp_mem, 1024);
    hipMemcpyAsync(output0, tmp_mem, 1024, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_6
// Description:	Constant
// Input:
// Output:
//	- name: Constant_6_0	type: float	shape: Shape{256}
void Constant_float_cuda_Constant_6(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_6_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_6_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1024];
    bin_file.read(tmp_mem, 1024);
    hipMemcpyAsync(output0, tmp_mem, 1024, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_93
// Description:	Constant
// Input:
// Output:
//	- name: Constant_93_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_93(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_93_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_93_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Greater_92
// Description:	Greater
// Input:
//	- name: Sum_90_0	type: int64_t	shape: Shape{}
//	- name: Constant_91_0	type: int64_t	shape: Shape{}
// Output:
//	- name: Greater_92_0	type: char	shape: Shape{}
extern "C" __launch_bounds__(1) __global__ void Greater_int64_t_int64_t_char_cuda_Greater_92(int64_t* input0, int64_t* input1, char* output0)
{
    output0[threadIdx.x] = greater(input0[0], input1[0]);

}
extern void Greater_int64_t_int64_t_char_cuda_Greater_92_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, int64_t* input0, int64_t* input1, char* output0) {
    Greater_int64_t_int64_t_char_cuda_Greater_92<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Node name:	Constant_54
// Description:	Constant
// Input:
// Output:
//	- name: Constant_54_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_54(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_54_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_54_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_88
// Description:	Constant
// Input:
// Output:
//	- name: Constant_88_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_88(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_88_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_88_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_11
// Description:	Constant
// Input:
// Output:
//	- name: Constant_11_0	type: float	shape: Shape{3797, 256}
void Constant_float_cuda_Constant_11(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_11_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_11_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[3888128];
    bin_file.read(tmp_mem, 3888128);
    hipMemcpyAsync(output0, tmp_mem, 3888128, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Result_98
// Description:	Result
// Input:
//	- name: tensor_87	type: int64_t	shape: Shape{50, 64}
// Output:
//	- name: Result_98_0	type: int64_t	shape: Shape{50, 64}
void Result_int64_t_int64_t_cuda_lib_Result_98(hipStream_t stream, int64_t* input0, int64_t* output0)
{
    if (input0 != output0)
        CUDA_SAFE_CALL(hipMemcpyAsync(output0, input0,25600, hipMemcpyDeviceToDevice, stream));
}
// Node name:	Constant_2
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2_0	type: float	shape: Shape{4, 256, 256}
void Constant_float_cuda_Constant_2(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1048576];
    bin_file.read(tmp_mem, 1048576);
    hipMemcpyAsync(output0, tmp_mem, 1048576, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Constant_12_0	type: float	shape: Shape{3797}
//	- name: Dot_77_0	type: float	shape: Shape{64, 3797}
//	- name: GatherV2_82_0	type: float	shape: Shape{64, 3797}
// Output:
//	- name: Add_83_0	type: float	shape: Shape{64, 3797}
// Fused functions:
// Broadcast, Broadcast_78
// Add, Add_79
// Reshape, @tmp_37
// Add, output_1
extern "C" __launch_bounds__(256) __global__ void FusedKernel_float_float_float_float_cuda_Broadcast_Add_Reshape_Add_1(float* input0, float* input1, float* input2, float* output0)
{
    for (int tid = blockIdx.x * 256 + threadIdx.x; tid < 243008; tid += 243200){
        float temp0 = input0[tid % 3797];
        float temp1 = add(input1[tid], temp0);
        float temp2 = add(temp1, input2[tid]);
        output0[tid] = temp2;
    }

}
extern void FusedKernel_float_float_float_float_cuda_Broadcast_Add_Reshape_Add_1_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0) {
    FusedKernel_float_float_float_float_cuda_Broadcast_Add_Reshape_Add_1<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0);
}
// Node name:	Constant_46
// Description:	Constant
// Input:
// Output:
//	- name: Constant_46_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_46(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_46_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_46_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_30
// Description:	Constant
// Input:
// Output:
//	- name: Constant_30_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_30(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_30_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_30_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_58
// Description:	Constant
// Input:
// Output:
//	- name: Constant_58_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_58(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_58_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_58_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_42
// Description:	Constant
// Input:
// Output:
//	- name: Constant_42_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_42(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_42_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_42_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_10
// Description:	Constant
// Input:
// Output:
//	- name: Constant_10_0	type: float	shape: Shape{256}
void Constant_float_cuda_Constant_10(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_10_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_10_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1024];
    bin_file.read(tmp_mem, 1024);
    hipMemcpyAsync(output0, tmp_mem, 1024, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_34
// Description:	Constant
// Input:
// Output:
//	- name: Constant_34_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_34(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_34_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_34_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_1
// Description:	Constant
// Input:
// Output:
//	- name: Constant_1_0	type: float	shape: Shape{4, 256, 256}
void Constant_float_cuda_Constant_1(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_1_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_1_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1048576];
    bin_file.read(tmp_mem, 1048576);
    hipMemcpyAsync(output0, tmp_mem, 1048576, hipMemcpyHostToDevice, stream);
    bin_file.close();

}

#ifndef __NNFUSION_GRAPH_CONFIG__
#define __NNFUSION_GRAPH_CONFIG__
#define NNFUSION_GRAPH_INPUT_NUM 6
#define NNFUSION_GRAPH_OUTPUT_NUM 6
#define NNFUSION_GRAPH_INPUT_DTYPE_0 float
#define NNFUSION_GRAPH_INPUT_SHAPE_0 {64, 256}
#define NNFUSION_GRAPH_INPUT_DTYPE_1 float
#define NNFUSION_GRAPH_INPUT_SHAPE_1 {64, 256}
#define NNFUSION_GRAPH_INPUT_DTYPE_2 float
#define NNFUSION_GRAPH_INPUT_SHAPE_2 {50, 64, 3797}
#define NNFUSION_GRAPH_INPUT_DTYPE_3 int64_t
#define NNFUSION_GRAPH_INPUT_SHAPE_3 {50, 64}
#define NNFUSION_GRAPH_INPUT_DTYPE_4 int64_t
#define NNFUSION_GRAPH_INPUT_SHAPE_4 {}
#define NNFUSION_GRAPH_INPUT_DTYPE_5 int64_t
#define NNFUSION_GRAPH_INPUT_SHAPE_5 {64}
#define NNFUSION_GRAPH_OUTPUT_DTYPE_0 float
#define NNFUSION_GRAPH_OUTPUT_SHAPE_0 {64, 256}
#define NNFUSION_GRAPH_OUTPUT_DTYPE_1 float
#define NNFUSION_GRAPH_OUTPUT_SHAPE_1 {64, 256}
#define NNFUSION_GRAPH_OUTPUT_DTYPE_2 int64_t
#define NNFUSION_GRAPH_OUTPUT_SHAPE_2 {50, 64}
#define NNFUSION_GRAPH_OUTPUT_DTYPE_3 char
#define NNFUSION_GRAPH_OUTPUT_SHAPE_3 {}
#define NNFUSION_GRAPH_OUTPUT_DTYPE_4 int64_t
#define NNFUSION_GRAPH_OUTPUT_SHAPE_4 {}
#define NNFUSION_GRAPH_OUTPUT_DTYPE_5 int64_t
#define NNFUSION_GRAPH_OUTPUT_SHAPE_5 {64}
#endif

// Node name:	Constant_38
// Description:	Constant
// Input:
// Output:
//	- name: Constant_38_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_38(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_38_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_38_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_91
// Description:	Constant
// Input:
// Output:
//	- name: Constant_91_0	type: int64_t	shape: Shape{}
void Constant_int64_t_cuda_Constant_91(hipStream_t stream, int64_t* output0)
{
    std::ifstream bin_file("./Constant/Constant_91_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_91_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[8];
    bin_file.read(tmp_mem, 8);
    hipMemcpyAsync(output0, tmp_mem, 8, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// 0: CUDA_GPU; 1: ROCM_GPU; 2: GENERIC_CPU; 3: HLSL; 4: GraphCore; 5: UNKNOWN
int get_device_type()
{
    return 0;
}

extern "C" void cuda_init()
{
//CUDA_SAFE_CALL(hipDeviceReset());
// total memory:14125248
CUDA_SAFE_CALL(hipSetDevice(0));
CUDA_SAFE_CALL(hipMalloc((void**)&group_0_CUDA_GPU0_allocator_memory_pool,4096000));
CUDA_SAFE_CALL(hipMemset((void*)group_0_CUDA_GPU0_allocator_memory_pool, 0, 4096000));
GatherV2_19_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+0);
GatherV2_82_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+65536);
Reshape_22_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+0);
Reshape_21_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1037568);
Reshape_26_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1037568);
BatchMatMul_24_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1103104);
BatchMatMul_28_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1365248);
Reshape_25_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1103104);
Reshape_29_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1365248);
GatherV2_55_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1627392);
GatherV2_47_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1692928);
GatherV2_39_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1758464);
GatherV2_31_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1824000);
GatherV2_59_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1889536);
GatherV2_51_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+1955072);
GatherV2_43_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+2020608);
GatherV2_35_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+2086144);
Dot_77_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+2151680);
Add_83_0 = (float*)(group_0_CUDA_GPU0_allocator_memory_pool+3123712);
Less_94_0 = (char*)(group_0_CUDA_GPU0_allocator_memory_pool+4095744);
Reshape_86_0 = (int64_t*)(group_0_CUDA_GPU0_allocator_memory_pool+4095808);
Sum_90_0 = (int64_t*)(group_0_CUDA_GPU0_allocator_memory_pool+4095872);
Greater_92_0 = (char*)(group_0_CUDA_GPU0_allocator_memory_pool+4095936);
CUDA_SAFE_CALL(hipSetDevice(0));
CUDA_SAFE_CALL(hipMalloc((void**)&group_persist_CUDA_GPU0_allocator_memory_pool,10029248));
CUDA_SAFE_CALL(hipMemset((void*)group_persist_CUDA_GPU0_allocator_memory_pool, 0, 10029248));
Constant_88_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+0);
Constant_0_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+64);
id_1 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+3888192);
Constant_11_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+3888256);
Constant_54_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+7776384);
Constant_1_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+7776448);
Reshape_23_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+7776448);
Constant_2_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+8825024);
Reshape_27_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+8825024);
Constant_46_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+9873600);
Constant_38_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+9873664);
Constant_30_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+9873728);
Constant_58_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+9873792);
Constant_50_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+9873856);
Constant_42_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+9873920);
Constant_34_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+9873984);
Constant_9_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9874048);
Constant_10_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9875072);
Constant_5_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9876096);
Constant_6_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9877120);
Constant_3_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9878144);
Constant_4_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9879168);
Constant_7_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9880192);
Constant_8_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9881216);
h_2 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9882240);
c_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9947776);
Reshape_76_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9882240);
Constant_12_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+10013312);
tensor_84 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+10028544);
Result_101_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+10028544);
Result_100_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+3888192);
Constant_93_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+10029056);
Constant_91_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+10029120);
// tensor_87 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+18446744073709551615);
cond_0 = (char*)(group_persist_CUDA_GPU0_allocator_memory_pool+10029184);
Result_99_0 = (char*)(group_persist_CUDA_GPU0_allocator_memory_pool+10029184);
// Result_98_0 = (int64_t*)(group_persist_CUDA_GPU0_allocator_memory_pool+18446744073709551615);
Result_97_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9947776);
Result_96_0 = (float*)(group_persist_CUDA_GPU0_allocator_memory_pool+9882240);
// create streams/handles
 // name=@tmp_42
Constant_int64_t_cuda_Constant_88(0, Constant_88_0);
 // name=embedding_0_weight
Constant_float_cuda_Constant_0(0, Constant_0_0);
 // name=out_0_weight
Constant_float_cuda_Constant_11(0, Constant_11_0);
 // name=@tmp_22
Constant_int64_t_cuda_Constant_54(0, Constant_54_0);
 // name=weight_ih_l0_t_0
Constant_float_cuda_Constant_1(0, Constant_1_0);
 // name=weight_hh_l0_t_0
Constant_float_cuda_Constant_2(0, Constant_2_0);
 // name=@tmp_16
Constant_int64_t_cuda_Constant_46(0, Constant_46_0);
 // name=@tmp_10
Constant_int64_t_cuda_Constant_38(0, Constant_38_0);
 // name=@tmp_4
Constant_int64_t_cuda_Constant_30(0, Constant_30_0);
 // name=@tmp_25
Constant_int64_t_cuda_Constant_58(0, Constant_58_0);
 // name=@tmp_19
Constant_int64_t_cuda_Constant_50(0, Constant_50_0);
 // name=@tmp_13
Constant_int64_t_cuda_Constant_42(0, Constant_42_0);
 // name=@tmp_7
Constant_int64_t_cuda_Constant_34(0, Constant_34_0);
 // name=bias_ih_3_0
Constant_float_cuda_Constant_9(0, Constant_9_0);
 // name=bias_hh_3_0
Constant_float_cuda_Constant_10(0, Constant_10_0);
 // name=bias_ih_1_0
Constant_float_cuda_Constant_5(0, Constant_5_0);
 // name=bias_hh_1_0
Constant_float_cuda_Constant_6(0, Constant_6_0);
 // name=bias_ih_0_0
Constant_float_cuda_Constant_3(0, Constant_3_0);
 // name=bias_hh_0_0
Constant_float_cuda_Constant_4(0, Constant_4_0);
 // name=bias_ih_2_0
Constant_float_cuda_Constant_7(0, Constant_7_0);
 // name=bias_hh_2_0
Constant_float_cuda_Constant_8(0, Constant_8_0);
 // name=out_0_bias
Constant_float_cuda_Constant_12(0, Constant_12_0);
 // name=@tmp_46
Constant_int64_t_cuda_Constant_93(0, Constant_93_0);
 // name=@tmp_44
Constant_int64_t_cuda_Constant_91(0, Constant_91_0);
CUDA_SAFE_CALL(hipDeviceGetAttribute(&num_SMs, hipDeviceAttributeMultiprocessorCount, 0));
}


extern "C" int kernel_entry(float* Parameter_13_0, float* Parameter_14_0, float* Parameter_15_0, int64_t* Parameter_16_0, int64_t* Parameter_17_0, int64_t* Parameter_18_0, float* Result_96_0, float* Result_97_0, int64_t* Result_98_0, char* Result_99_0, int64_t* Result_100_0, int64_t* Result_101_0)
{
// kernel_entry_init
 // name=blockfusion_kernel_104
BlockFusionKernel_float_int64_t_float_int64_t_int64_t_float_float_int64_t_cuda_GatherV2_GatherV2_Add_0_Call(dim3(1015, 1, 1), dim3(256, 1, 1), 0, 0, Parameter_15_0, Parameter_17_0, Constant_0_0, Parameter_18_0, Constant_88_0, GatherV2_82_0, GatherV2_19_0, Result_100_0);
 // name=Reshape_23
// eliminated: Reshape_float_float_cuda_Reshape_23_Call(dim3(1024, 1, 1), dim3(256, 1, 1), 0, 0, Constant_1_0, Reshape_23_0);
 // name=Reshape_22
// eliminated: Reshape_float_float_cuda_Reshape_22_Call(dim3(64, 1, 1), dim3(256, 1, 1), 0, 0, GatherV2_19_0, Reshape_22_0);
 // name=Reshape_27
// eliminated: Reshape_float_float_cuda_Reshape_27_Call(dim3(1024, 1, 1), dim3(256, 1, 1), 0, 0, Constant_2_0, Reshape_27_0);
 // name=h_0
Reshape_float_float_cuda_Reshape_21_Call(dim3(64, 1, 1), dim3(256, 1, 1), 0, 0, Parameter_13_0, Reshape_21_0);
 // name=Reshape_26
// eliminated: Reshape_float_float_cuda_Reshape_21_Call(dim3(64, 1, 1), dim3(256, 1, 1), 0, 0, Reshape_21_0, Reshape_26_0);
 // name=blockfusion_kernel_105
BlockFusionKernel_float_float_float_float_float_float_cuda_BatchMatMul_BatchMatMul_1_Call(dim3(128, 1, 1), dim3(256, 1, 1), 0, 0, Reshape_26_0, Reshape_27_0, Reshape_22_0, Reshape_23_0, BatchMatMul_28_0, BatchMatMul_24_0);
 // name=Reshape_25
// eliminated: Reshape_float_float_cuda_Reshape_25_Call(dim3(256, 1, 1), dim3(256, 1, 1), 0, 0, BatchMatMul_24_0, Reshape_25_0);
 // name=Reshape_29
// eliminated: Reshape_float_float_cuda_Reshape_29_Call(dim3(256, 1, 1), dim3(256, 1, 1), 0, 0, BatchMatMul_28_0, Reshape_29_0);
 // name=blockfusion_kernel_106
BlockFusionKernel_float_int64_t_int64_t_int64_t_int64_t_float_int64_t_int64_t_int64_t_int64_t_float_float_float_float_float_float_float_float_cuda_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_GatherV2_2_Call(dim3(512, 1, 1), dim3(256, 1, 1), 0, 0, Reshape_29_0, Constant_34_0, Constant_42_0, Constant_50_0, Constant_58_0, Reshape_25_0, Constant_30_0, Constant_38_0, Constant_46_0, Constant_54_0, GatherV2_35_0, GatherV2_43_0, GatherV2_51_0, GatherV2_59_0, GatherV2_31_0, GatherV2_39_0, GatherV2_47_0, GatherV2_55_0);
 // name=ElementWiseFused_102
FusedKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Broadcast_Add_Broadcast_Add_Add_Tanh_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_Add_Tanh_Broadcast_Add_Broadcast_Add_Add_Sigmoid_Multiply_0_Call(dim3(64, 1, 1), dim3(256, 1, 1), 0, 0, Constant_8_0, GatherV2_51_0, Constant_7_0, GatherV2_47_0, Constant_4_0, GatherV2_35_0, Constant_3_0, GatherV2_31_0, Constant_6_0, GatherV2_43_0, Constant_5_0, GatherV2_39_0, Parameter_14_0, Constant_10_0, GatherV2_59_0, Constant_9_0, GatherV2_55_0, Result_97_0, Result_96_0);
 // name=@tmp_33
Reshape_float_float_cuda_Reshape_21_Call(dim3(64, 1, 1), dim3(256, 1, 1), 0, 0, Result_96_0, Reshape_76_0);
 // name=Dot_77
Dot_float_float_float_cuda_Dot_77_Call(dim3(80, 1, 1), dim3(384, 1, 1), 0, 0, Reshape_76_0, Constant_11_0, Dot_77_0);
 // name=ElementWiseFused_103
FusedKernel_float_float_float_float_cuda_Broadcast_Add_Reshape_Add_1_Call(dim3(950, 1, 1), dim3(256, 1, 1), 0, 0, Constant_12_0, Dot_77_0, GatherV2_82_0, Add_83_0);
 // name=output_2
ArgMax_float_int64_t_cuda_ArgMax_84_Call(dim3(64, 1, 1), dim3(256, 1, 1), 0, 0, Add_83_0, Result_101_0);
 // name=Result_101
// eliminated (extern_result_memory): Result_int64_t_int64_t_cuda_lib_Result_101(0, Result_101_0, Result_101_0);
 // name=Result_100
// eliminated (extern_result_memory): Result_int64_t_int64_t_cuda_lib_Result_100(0, Result_100_0, Result_100_0);
 // name=@tmp_47
Less_int64_t_int64_t_char_cuda_Less_94_Call(dim3(1, 1, 1), dim3(1, 1, 1), 0, 0, Result_100_0, Constant_93_0, Less_94_0);
 // name=@tmp_40
Reshape_int64_t_int64_t_cuda_Reshape_86_Call(dim3(1, 1, 1), dim3(256, 1, 1), 0, 0, Parameter_17_0, Reshape_86_0);
 // name=blockfusion_kernel_107
tensor_87 = Parameter_16_0;
/* memref */BlockFusionKernel_int64_t_int64_t_int64_t_int64_t_int64_t_cuda_ScatterND_Sum_3_Call(dim3(2, 1, 1), dim3(256, 1, 1), 0, 0, Parameter_16_0, Reshape_86_0, Result_101_0, Result_98_0, Sum_90_0);
 // name=@tmp_45
Greater_int64_t_int64_t_char_cuda_Greater_92_Call(dim3(1, 1, 1), dim3(1, 1, 1), 0, 0, Sum_90_0, Constant_91_0, Greater_92_0);
 // name=cond_0
And_char_char_char_cuda_And_95_Call(dim3(1, 1, 1), dim3(1, 1, 1), 0, 0, Greater_92_0, Less_94_0, Result_99_0);
 // name=Result_99
// eliminated (extern_result_memory): Result_char_char_cuda_lib_Result_99(0, Result_99_0, Result_99_0);
 // name=Result_98
// eliminated (extern_result_memory): Result_int64_t_int64_t_cuda_lib_Result_98(0, Result_98_0, Result_98_0);
 // name=Result_97
// eliminated (extern_result_memory): Result_float_float_cuda_lib_Result_97(0, Result_97_0, Result_97_0);
 // name=Result_96
// eliminated (extern_result_memory): Result_float_float_cuda_lib_Result_97(0, Result_96_0, Result_96_0);
return 0;
}


extern "C" void cuda_free()
{
CUDA_SAFE_CALL(hipSetDevice(0));
CUDA_SAFE_CALL(hipFree(group_0_CUDA_GPU0_allocator_memory_pool));
CUDA_SAFE_CALL(hipSetDevice(0));
CUDA_SAFE_CALL(hipFree(group_persist_CUDA_GPU0_allocator_memory_pool));
}

// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

