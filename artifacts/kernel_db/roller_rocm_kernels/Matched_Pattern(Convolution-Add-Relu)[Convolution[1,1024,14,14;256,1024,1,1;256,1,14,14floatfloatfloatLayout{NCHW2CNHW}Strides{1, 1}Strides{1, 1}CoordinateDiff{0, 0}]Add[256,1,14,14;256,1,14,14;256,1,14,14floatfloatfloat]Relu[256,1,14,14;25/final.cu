#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[1];
  __shared__ float data_pad_shared[256];
  __shared__ float kernel_pad_shared[512];
  float data_pad_shared_local[1];
  float kernel_pad_shared_local[1];
  conv_local[0] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 32; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[((int)threadIdx.x)] = (((((((int)blockIdx.x) % 25) * 2) + ((((int)threadIdx.x) & 7) >> 2)) < 49) ? data[((((ra_fused0_outer * 6272) + ((((int)threadIdx.x) >> 3) * 196)) + ((((int)blockIdx.x) % 25) * 8)) + (((int)threadIdx.x) & 7))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 128)] = (((((((int)blockIdx.x) % 25) * 2) + ((((int)threadIdx.x) & 7) >> 2)) < 49) ? data[(((((ra_fused0_outer * 6272) + ((((int)threadIdx.x) >> 3) * 196)) + ((((int)blockIdx.x) % 25) * 8)) + (((int)threadIdx.x) & 7)) + 3136)] : 0.000000e+00f);
    kernel_pad_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 25) * 16384) + ((((int)threadIdx.x) >> 5) * 1024)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_pad_shared[(((int)threadIdx.x) + 128)] = kernel[((((((((int)blockIdx.x) / 25) * 16384) + ((((int)threadIdx.x) >> 5) * 1024)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    kernel_pad_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)blockIdx.x) / 25) * 16384) + ((((int)threadIdx.x) >> 5) * 1024)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192)];
    kernel_pad_shared[(((int)threadIdx.x) + 384)] = kernel[((((((((int)blockIdx.x) / 25) * 16384) + ((((int)threadIdx.x) >> 5) * 1024)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288)];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[0] = data_pad_shared[((ra_fused0_inner_outer * 8) + (((int)threadIdx.x) & 7))];
      kernel_pad_shared_local[0] = kernel_pad_shared[(((((int)threadIdx.x) >> 3) * 32) + ra_fused0_inner_outer)];
      conv_local[0] = (conv_local[0] + (data_pad_shared_local[0] * kernel_pad_shared_local[0]));
    }
  }
  if ((((((int)blockIdx.x) % 25) * 2) + ((((int)threadIdx.x) & 7) >> 2)) < 49) {
    conv_unpad[(((((((int)blockIdx.x) / 25) * 3136) + ((((int)threadIdx.x) >> 3) * 196)) + ((((int)blockIdx.x) % 25) * 8)) + (((int)threadIdx.x) & 7))] = max((conv_local[0] + bias[(((((int)blockIdx.x) / 25) * 16) + (((int)threadIdx.x) >> 3))]), 0.000000e+00f);
  }
}

dim3 grid(400, 1, 1);
dim3 block(128, 1, 1);
best_idx 5