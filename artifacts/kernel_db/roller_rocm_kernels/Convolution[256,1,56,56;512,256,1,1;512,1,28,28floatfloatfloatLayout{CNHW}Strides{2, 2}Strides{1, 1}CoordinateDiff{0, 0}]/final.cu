#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv_unpad) {
  float conv_local[8];
  __shared__ float data_pad_shared[1024];
  __shared__ float kernel_pad_shared[512];
  float data_pad_shared_local[4];
  float kernel_pad_shared_local[2];
  conv_local[0] = 0.000000e+00f;
  conv_local[4] = 0.000000e+00f;
  conv_local[1] = 0.000000e+00f;
  conv_local[5] = 0.000000e+00f;
  conv_local[2] = 0.000000e+00f;
  conv_local[6] = 0.000000e+00f;
  conv_local[3] = 0.000000e+00f;
  conv_local[7] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 16; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[((int)threadIdx.x)] = (((((((int)blockIdx.x) % 13) * 4) + ((((int)threadIdx.x) & 63) >> 4)) < 49) ? data[((((ra_fused0_outer * 50176) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((((int)blockIdx.x) % 13) * 16) + ((((int)threadIdx.x) & 63) >> 2)) / 7) * 112)) + (((((((int)blockIdx.x) % 13) * 8) + (((int)threadIdx.x) & 63)) % 28) * 2))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 256)] = (((((((int)blockIdx.x) % 13) * 4) + ((((int)threadIdx.x) & 63) >> 4)) < 49) ? data[(((((ra_fused0_outer * 50176) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((((int)blockIdx.x) % 13) * 16) + ((((int)threadIdx.x) & 63) >> 2)) / 7) * 112)) + (((((((int)blockIdx.x) % 13) * 8) + (((int)threadIdx.x) & 63)) % 28) * 2)) + 12544)] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 512)] = (((((((int)blockIdx.x) % 13) * 4) + ((((int)threadIdx.x) & 63) >> 4)) < 49) ? data[(((((ra_fused0_outer * 50176) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((((int)blockIdx.x) % 13) * 16) + ((((int)threadIdx.x) & 63) >> 2)) / 7) * 112)) + (((((((int)blockIdx.x) % 13) * 8) + (((int)threadIdx.x) & 63)) % 28) * 2)) + 25088)] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 768)] = (((((((int)blockIdx.x) % 13) * 4) + ((((int)threadIdx.x) & 63) >> 4)) < 49) ? data[(((((ra_fused0_outer * 50176) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((((int)blockIdx.x) % 13) * 16) + ((((int)threadIdx.x) & 63) >> 2)) / 7) * 112)) + (((((((int)blockIdx.x) % 13) * 8) + (((int)threadIdx.x) & 63)) % 28) * 2)) + 37632)] : 0.000000e+00f);
    kernel_pad_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 13) * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + (ra_fused0_outer * 16)) + (((int)threadIdx.x) & 15))];
    kernel_pad_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)blockIdx.x) / 13) * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + (ra_fused0_outer * 16)) + (((int)threadIdx.x) & 15)) + 4096)];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 16; ++ra_fused0_inner_outer) {
      data_pad_shared_local[0] = data_pad_shared[((ra_fused0_inner_outer * 64) + (((int)threadIdx.x) & 15))];
      data_pad_shared_local[1] = data_pad_shared[(((ra_fused0_inner_outer * 64) + (((int)threadIdx.x) & 15)) + 16)];
      data_pad_shared_local[2] = data_pad_shared[(((ra_fused0_inner_outer * 64) + (((int)threadIdx.x) & 15)) + 32)];
      data_pad_shared_local[3] = data_pad_shared[(((ra_fused0_inner_outer * 64) + (((int)threadIdx.x) & 15)) + 48)];
      kernel_pad_shared_local[0] = kernel_pad_shared[(((((int)threadIdx.x) >> 4) * 16) + ra_fused0_inner_outer)];
      kernel_pad_shared_local[1] = kernel_pad_shared[((((((int)threadIdx.x) >> 4) * 16) + ra_fused0_inner_outer) + 256)];
      conv_local[0] = (conv_local[0] + (data_pad_shared_local[0] * kernel_pad_shared_local[0]));
      conv_local[4] = (conv_local[4] + (data_pad_shared_local[0] * kernel_pad_shared_local[1]));
      conv_local[1] = (conv_local[1] + (data_pad_shared_local[1] * kernel_pad_shared_local[0]));
      conv_local[5] = (conv_local[5] + (data_pad_shared_local[1] * kernel_pad_shared_local[1]));
      conv_local[2] = (conv_local[2] + (data_pad_shared_local[2] * kernel_pad_shared_local[0]));
      conv_local[6] = (conv_local[6] + (data_pad_shared_local[2] * kernel_pad_shared_local[1]));
      conv_local[3] = (conv_local[3] + (data_pad_shared_local[3] * kernel_pad_shared_local[0]));
      conv_local[7] = (conv_local[7] + (data_pad_shared_local[3] * kernel_pad_shared_local[1]));
    }
  }
  conv_unpad[(((((((int)blockIdx.x) / 13) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 13) * 64)) + (((int)threadIdx.x) & 15))] = conv_local[0];
  conv_unpad[((((((((int)blockIdx.x) / 13) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 13) * 64)) + (((int)threadIdx.x) & 15)) + 12544)] = conv_local[4];
  if ((((int)blockIdx.x) % 13) < 12) {
    conv_unpad[((((((((int)blockIdx.x) / 13) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 13) * 64)) + (((int)threadIdx.x) & 15)) + 16)] = conv_local[1];
    conv_unpad[((((((((int)blockIdx.x) / 13) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 13) * 64)) + (((int)threadIdx.x) & 15)) + 12560)] = conv_local[5];
  }
  if ((((int)blockIdx.x) % 13) < 12) {
    conv_unpad[((((((((int)blockIdx.x) / 13) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 13) * 64)) + (((int)threadIdx.x) & 15)) + 32)] = conv_local[2];
    conv_unpad[((((((((int)blockIdx.x) / 13) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 13) * 64)) + (((int)threadIdx.x) & 15)) + 12576)] = conv_local[6];
  }
  if ((((int)blockIdx.x) % 13) < 12) {
    conv_unpad[((((((((int)blockIdx.x) / 13) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 13) * 64)) + (((int)threadIdx.x) & 15)) + 48)] = conv_local[3];
    conv_unpad[((((((((int)blockIdx.x) / 13) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 13) * 64)) + (((int)threadIdx.x) & 15)) + 12592)] = conv_local[7];
  }
}

dim3 grid(208, 1, 1);
dim3 block(256, 1, 1);
best_idx 8