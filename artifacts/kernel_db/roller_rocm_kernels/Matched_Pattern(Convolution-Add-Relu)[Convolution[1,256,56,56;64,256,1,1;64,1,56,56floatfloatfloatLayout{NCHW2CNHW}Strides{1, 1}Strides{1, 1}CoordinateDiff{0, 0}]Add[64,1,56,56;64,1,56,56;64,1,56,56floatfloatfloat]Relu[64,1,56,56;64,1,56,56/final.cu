#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[2];
  __shared__ float data_pad_shared[512];
  __shared__ float kernel_pad_shared[1024];
  float data_pad_shared_local[2];
  float kernel_pad_shared_local[1];
  conv_local[0] = 0.000000e+00f;
  conv_local[1] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 8; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[((int)threadIdx.x)] = data[((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 4) * 3136)) + ((((int)blockIdx.x) % 196) * 16)) + (((int)threadIdx.x) & 15))];
    data_pad_shared[(((int)threadIdx.x) + 256)] = data[(((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 4) * 3136)) + ((((int)blockIdx.x) % 196) * 16)) + (((int)threadIdx.x) & 15)) + 50176)];
    kernel_pad_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 196) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_pad_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)blockIdx.x) / 196) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048)];
    kernel_pad_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)blockIdx.x) / 196) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    kernel_pad_shared[(((int)threadIdx.x) + 768)] = kernel[((((((((int)blockIdx.x) / 196) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144)];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[0] = data_pad_shared[((ra_fused0_inner_outer * 16) + (((int)threadIdx.x) & 7))];
      data_pad_shared_local[1] = data_pad_shared[(((ra_fused0_inner_outer * 16) + (((int)threadIdx.x) & 7)) + 8)];
      kernel_pad_shared_local[0] = kernel_pad_shared[(((((int)threadIdx.x) >> 3) * 32) + ra_fused0_inner_outer)];
      conv_local[0] = (conv_local[0] + (data_pad_shared_local[0] * kernel_pad_shared_local[0]));
      conv_local[1] = (conv_local[1] + (data_pad_shared_local[1] * kernel_pad_shared_local[0]));
    }
  }
  conv_unpad[(((((((int)blockIdx.x) / 196) * 100352) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 196) * 16)) + (((int)threadIdx.x) & 7))] = max((conv_local[0] + bias[(((((int)blockIdx.x) / 196) * 32) + (((int)threadIdx.x) >> 3))]), 0.000000e+00f);
  conv_unpad[((((((((int)blockIdx.x) / 196) * 100352) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 196) * 16)) + (((int)threadIdx.x) & 7)) + 8)] = max((conv_local[1] + bias[(((((int)blockIdx.x) / 196) * 32) + (((int)threadIdx.x) >> 3))]), 0.000000e+00f);
}

dim3 grid(392, 1, 1);
dim3 block(256, 1, 1);
best_idx 6