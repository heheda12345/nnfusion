#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[2];
  __shared__ float data_pad_shared[256];
  __shared__ float kernel_pad_shared[512];
  float data_pad_shared_local[1];
  float kernel_pad_shared_local[2];
  conv_local[0] = 0.000000e+00f;
  conv_local[1] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 72; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[((int)threadIdx.x)] = (((0 < ((((((((int)blockIdx.x) % 49) * 4) + ((((int)threadIdx.x) & 15) >> 2)) / 7) * 2) + ((((ra_fused0_outer * 7) + (((int)threadIdx.x) >> 4)) % 9) / 3))) && (0 < ((((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) % 28) * 2) + (((((int)threadIdx.x) >> 4) + ra_fused0_outer) % 3)))) ? data[(((((((((ra_fused0_outer * 16) + (((int)threadIdx.x) >> 4)) / 9) * 3136) + (((((((int)blockIdx.x) % 49) * 4) + ((((int)threadIdx.x) & 15) >> 2)) / 7) * 112)) + (((((ra_fused0_outer * 7) + (((int)threadIdx.x) >> 4)) % 9) / 3) * 56)) + (((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) % 28) * 2)) + (((((int)threadIdx.x) >> 4) + ra_fused0_outer) % 3)) - 57)] : 0.000000e+00f);
    kernel_pad_shared[((int)threadIdx.x)] = kernel[((((((((int)blockIdx.x) / 49) * 36864) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((ra_fused0_outer * 16) + (((int)threadIdx.x) & 15)) / 9) * 9)) + (((((ra_fused0_outer * 7) + (((int)threadIdx.x) & 15)) % 9) / 3) * 3)) + ((ra_fused0_outer + (((int)threadIdx.x) & 15)) % 3))];
    kernel_pad_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((((int)blockIdx.x) / 49) * 36864) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((ra_fused0_outer * 16) + (((int)threadIdx.x) & 15)) / 9) * 9)) + (((((ra_fused0_outer * 7) + (((int)threadIdx.x) & 15)) % 9) / 3) * 3)) + ((ra_fused0_outer + (((int)threadIdx.x) & 15)) % 3)) + 18432)];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 16; ++ra_fused0_inner_outer) {
      data_pad_shared_local[0] = data_pad_shared[((ra_fused0_inner_outer * 16) + (((int)threadIdx.x) & 15))];
      kernel_pad_shared_local[0] = kernel_pad_shared[(((((int)threadIdx.x) >> 4) * 16) + ra_fused0_inner_outer)];
      kernel_pad_shared_local[1] = kernel_pad_shared[((((((int)threadIdx.x) >> 4) * 16) + ra_fused0_inner_outer) + 256)];
      conv_local[0] = (conv_local[0] + (data_pad_shared_local[0] * kernel_pad_shared_local[0]));
      conv_local[1] = (conv_local[1] + (data_pad_shared_local[0] * kernel_pad_shared_local[1]));
    }
  }
  conv_unpad[(((((((int)blockIdx.x) / 49) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 49) * 16)) + (((int)threadIdx.x) & 15))] = max((conv_local[0] + bias[(((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 4))]), 0.000000e+00f);
  conv_unpad[((((((((int)blockIdx.x) / 49) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 49) * 16)) + (((int)threadIdx.x) & 15)) + 12544)] = max((conv_local[1] + bias[((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 4)) + 16)]), 0.000000e+00f);
}

dim3 grid(196, 1, 1);
dim3 block(256, 1, 1);
best_idx 0