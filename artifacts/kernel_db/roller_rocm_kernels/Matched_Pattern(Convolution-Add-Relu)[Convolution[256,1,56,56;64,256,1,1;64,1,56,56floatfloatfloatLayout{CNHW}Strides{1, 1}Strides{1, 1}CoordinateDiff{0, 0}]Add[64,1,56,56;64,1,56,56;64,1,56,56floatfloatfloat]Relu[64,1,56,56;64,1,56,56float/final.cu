#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[2];
  __shared__ float data_pad_shared[512];
  __shared__ float kernel_pad_shared[256];
  float data_pad_shared_local[1];
  float kernel_pad_shared_local[2];
  conv_local[0] = 0.000000e+00f;
  conv_local[1] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 16; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[((int)threadIdx.x)] = data[((((ra_fused0_outer * 50176) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 98) * 32)) + (((int)threadIdx.x) & 31))];
    data_pad_shared[(((int)threadIdx.x) + 256)] = data[(((((ra_fused0_outer * 50176) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 98) * 32)) + (((int)threadIdx.x) & 31)) + 25088)];
    kernel_pad_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 98) * 4096) + ((((int)threadIdx.x) >> 4) * 256)) + (ra_fused0_outer * 16)) + (((int)threadIdx.x) & 15))];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 16; ++ra_fused0_inner_outer) {
      data_pad_shared_local[0] = data_pad_shared[((ra_fused0_inner_outer * 32) + (((int)threadIdx.x) & 31))];
      kernel_pad_shared_local[0] = kernel_pad_shared[(((((int)threadIdx.x) >> 5) * 16) + ra_fused0_inner_outer)];
      kernel_pad_shared_local[1] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 16) + ra_fused0_inner_outer) + 128)];
      conv_local[0] = (conv_local[0] + (data_pad_shared_local[0] * kernel_pad_shared_local[0]));
      conv_local[1] = (conv_local[1] + (data_pad_shared_local[0] * kernel_pad_shared_local[1]));
    }
  }
  conv_unpad[(((((((int)blockIdx.x) / 98) * 50176) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 98) * 32)) + (((int)threadIdx.x) & 31))] = max((conv_local[0] + bias[(((((int)blockIdx.x) / 98) * 16) + (((int)threadIdx.x) >> 5))]), 0.000000e+00f);
  conv_unpad[((((((((int)blockIdx.x) / 98) * 50176) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 98) * 32)) + (((int)threadIdx.x) & 31)) + 25088)] = max((conv_local[1] + bias[((((((int)blockIdx.x) / 98) * 16) + (((int)threadIdx.x) >> 5)) + 8)]), 0.000000e+00f);
}

dim3 grid(392, 1, 1);
dim3 block(256, 1, 1);
best_idx 1