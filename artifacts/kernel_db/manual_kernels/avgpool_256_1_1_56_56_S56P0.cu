#include <hip/hip_runtime.h>
#include <stdexcept>
#include <sstream>

#define CUDA_SAFE_CALL(x)                                                                          \
    do                                                                                             \
    {                                                                                              \
        hipError_t result = (x);                                                                  \
        if (result != hipSuccess)                                                                 \
        {                                                                                          \
            const char* msg = hipGetErrorString(result);                                          \
            std::stringstream safe_call_ss;                                                        \
            safe_call_ss << "\nerror: " #x " failed with error"                                    \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg;  \
            throw std::runtime_error(safe_call_ss.str());                                          \
        }                                                                                          \
    } while (0)

// %%%
extern "C" __global__ void default_function_kernel0(float* __restrict__ input0, float* __restrict__ output0) {
    int start_id = blockIdx.x * 56 * 56;
    int end_id = (blockIdx.x + 1) * 56 * 56;
    int tid = threadIdx.x;
    float sum = 0;
    for (int i = start_id + tid; i < end_id; i += blockDim.x) {
        sum += input0[i];
    }
    __shared__ float share_sum[256];
    share_sum[tid] = sum;
    __syncthreads();
    if (tid < 128) share_sum[tid] += share_sum[tid + 128]; __syncthreads();
    if (tid < 64) share_sum[tid] += share_sum[tid + 64]; __syncthreads();
    if (tid < 32) {
        share_sum[tid] += share_sum[tid + 32];
        float s = share_sum[tid];
        s += __shfl_xor_sync(0xffffffff, s, 16);
        s += __shfl_xor_sync(0xffffffff, s, 8);
        s += __shfl_xor_sync(0xffffffff, s, 4);
        s += __shfl_xor_sync(0xffffffff, s, 2);
        s += __shfl_xor_sync(0xffffffff, s, 1);
        if (tid == 0) output0[blockIdx.x] = s * 0.00031887755102040814; // s / 56 / 56
    }
}
// %%%

// +++
dim3 grid(256, 1, 1);
dim3 block(256, 1, 1);
// +++

int main() {
    float *input0, *output0;
    hipMallocManaged(&input0, 256 * 56 * 56 * sizeof(float));
    hipMallocManaged(&output0, 256 * sizeof(float));
    for (int i = 0; i < 256 * 56 * 56; i++) input0[i] = i;
    default_function_kernel0<<<grid, block>>>(input0, output0);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    for (int i = 0; i < 256; i++) printf("%f ", output0[i]);
    return 0;
}

// python
// s = list(range(0, 56 * 56)) 
// ans = [sum([3136 * i + x for x in s]) / 56 / 56 for i in range(256)]
// print(ans)