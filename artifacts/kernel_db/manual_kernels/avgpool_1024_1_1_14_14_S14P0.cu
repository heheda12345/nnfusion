#include <hip/hip_runtime.h>
#include <stdexcept>
#include <sstream>

#define CUDA_SAFE_CALL(x)                                                                          \
    do                                                                                             \
    {                                                                                              \
        hipError_t result = (x);                                                                  \
        if (result != hipSuccess)                                                                 \
        {                                                                                          \
            const char* msg = hipGetErrorString(result);                                          \
            std::stringstream safe_call_ss;                                                        \
            safe_call_ss << "\nerror: " #x " failed with error"                                    \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg;  \
            throw std::runtime_error(safe_call_ss.str());                                          \
        }                                                                                          \
    } while (0)

// %%%
extern "C" __global__ void default_function_kernel0(float* __restrict__ input0, float* __restrict__ output0) {
    int start_id = blockIdx.x * 14 * 14;
    int tid = threadIdx.x;
    float sum = tid < 14 * 14 ? input0[start_id + tid] : 0;
    __shared__ float share_sum[224];
    share_sum[tid] = sum;
    __syncthreads();
    if (tid < 96) share_sum[tid] += share_sum[tid + 128]; __syncthreads();
    if (tid < 64) share_sum[tid] += share_sum[tid + 64]; __syncthreads();
    if (tid < 32) {
        share_sum[tid] += share_sum[tid + 32];
        float s = share_sum[tid];
        s += __shfl_xor_sync(0xffffffff, s, 16);
        s += __shfl_xor_sync(0xffffffff, s, 8);
        s += __shfl_xor_sync(0xffffffff, s, 4);
        s += __shfl_xor_sync(0xffffffff, s, 2);
        s += __shfl_xor_sync(0xffffffff, s, 1);
        if (tid == 0) output0[blockIdx.x] = s * 0.00510204081632653; // s / 14 / 14
    }
}
// %%%

// +++
dim3 grid(1024, 1, 1);
dim3 block(224, 1, 1);
// +++

int main() {
    float *input0, *output0;
    hipMallocManaged(&input0, 1024 * 14 * 14 * sizeof(float));
    hipMallocManaged(&output0, 1024 * sizeof(float));
    for (int i = 0; i < 1024 * 14 * 14; i++) input0[i] = i;
    default_function_kernel0<<<grid, block>>>(input0, output0);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    for (int i = 0; i < 1024; i++) printf("%f ", output0[i]);
    return 0;
}

// python
// s = list(range(0, 14 * 14)) 
// print([sum([14 * 14 * i + x for x in s]) / 14 / 14 for i in range(1024)])
