#include <hip/hip_runtime.h>
#include <stdexcept>
#include <sstream>

#define CUDA_SAFE_CALL(x)                                                                          \
    do                                                                                             \
    {                                                                                              \
        hipError_t result = (x);                                                                  \
        if (result != hipSuccess)                                                                 \
        {                                                                                          \
            const char* msg = hipGetErrorString(result);                                          \
            std::stringstream safe_call_ss;                                                        \
            safe_call_ss << "\nerror: " #x " failed with error"                                    \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg;  \
            throw std::runtime_error(safe_call_ss.str());                                          \
        }                                                                                          \
    } while (0)

// %%%
extern "C" __global__ void default_function_kernel0(float* __restrict__ input0, float* __restrict__ input1, float* __restrict__ output0) {
    int lane_id = threadIdx.x >> 5;
    int tid = threadIdx.x & 31;
    int task_id = blockIdx.x * 8 + lane_id;
    if (task_id < 10) {
        float s = input0[tid] * input1[task_id * 64 + tid] + input0[tid + 32] * input1[task_id * 64 + tid + 32];
        s += __shfl_xor_sync(0xffffffff, s, 16);
        s += __shfl_xor_sync(0xffffffff, s, 8);
        s += __shfl_xor_sync(0xffffffff, s, 4);
        s += __shfl_xor_sync(0xffffffff, s, 2);
        s += __shfl_xor_sync(0xffffffff, s, 1);
        if (tid == 0) output0[task_id] = s;
    }
}
// %%%

// +++
dim3 grid(2, 1, 1);
dim3 block(256, 1, 1);
// +++

int main() {
    float *input0, *input1, *output0;
    hipMallocManaged(&input0, 64 * sizeof(float));
    hipMallocManaged(&input1, 64 * 10 * sizeof(float));
    hipMallocManaged(&output0, 64 * 10 * sizeof(float));
    for (int i = 0; i < 64; i++) input0[i] = i;
    for (int i = 0; i < 64 * 10; i++) input1[i] = i;
    default_function_kernel0<<<grid, block>>>(input0, input1, output0);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    for (int i = 0; i < 10; i++) printf("%f ", output0[i]);
    return 0;
}

// answer
// s = list(range(0, 64))
// for i in range(10): print(sum([x * (64 * i + x) for x in s]))
// 85344
// 214368
// 343392
// 472416
// 601440
// 730464
// 859488
// 988512
// 1117536
// 1246560