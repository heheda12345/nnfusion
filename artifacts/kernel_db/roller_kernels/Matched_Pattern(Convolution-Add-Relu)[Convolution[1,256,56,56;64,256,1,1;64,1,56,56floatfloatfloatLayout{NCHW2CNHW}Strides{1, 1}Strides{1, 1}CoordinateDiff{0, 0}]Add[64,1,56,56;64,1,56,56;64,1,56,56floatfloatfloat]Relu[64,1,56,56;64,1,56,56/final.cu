#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[16];
  __shared__ float data_pad_shared[2048];
  __shared__ float kernel_pad_shared[1024];
  float data_pad_shared_local[2];
  float kernel_pad_shared_local[8];
  conv_local[(0)] = 0.000000e+00f;
  conv_local[(2)] = 0.000000e+00f;
  conv_local[(4)] = 0.000000e+00f;
  conv_local[(6)] = 0.000000e+00f;
  conv_local[(8)] = 0.000000e+00f;
  conv_local[(10)] = 0.000000e+00f;
  conv_local[(12)] = 0.000000e+00f;
  conv_local[(14)] = 0.000000e+00f;
  conv_local[(1)] = 0.000000e+00f;
  conv_local[(3)] = 0.000000e+00f;
  conv_local[(5)] = 0.000000e+00f;
  conv_local[(7)] = 0.000000e+00f;
  conv_local[(9)] = 0.000000e+00f;
  conv_local[(11)] = 0.000000e+00f;
  conv_local[(13)] = 0.000000e+00f;
  conv_local[(15)] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 8; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[(((int)threadIdx.x))] = data[(((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)))];
    data_pad_shared[((((int)threadIdx.x) + 128))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 6272))];
    data_pad_shared[((((int)threadIdx.x) + 256))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 12544))];
    data_pad_shared[((((int)threadIdx.x) + 384))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 18816))];
    data_pad_shared[((((int)threadIdx.x) + 512))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 25088))];
    data_pad_shared[((((int)threadIdx.x) + 640))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 31360))];
    data_pad_shared[((((int)threadIdx.x) + 768))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 37632))];
    data_pad_shared[((((int)threadIdx.x) + 896))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 43904))];
    data_pad_shared[((((int)threadIdx.x) + 1024))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 50176))];
    data_pad_shared[((((int)threadIdx.x) + 1152))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 56448))];
    data_pad_shared[((((int)threadIdx.x) + 1280))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 62720))];
    data_pad_shared[((((int)threadIdx.x) + 1408))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 68992))];
    data_pad_shared[((((int)threadIdx.x) + 1536))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 75264))];
    data_pad_shared[((((int)threadIdx.x) + 1664))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 81536))];
    data_pad_shared[((((int)threadIdx.x) + 1792))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 87808))];
    data_pad_shared[((((int)threadIdx.x) + 1920))] = data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 63)) + 94080))];
    kernel_pad_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)))];
    kernel_pad_shared[((((int)threadIdx.x) + 128))] = kernel[(((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 1024))];
    kernel_pad_shared[((((int)threadIdx.x) + 256))] = kernel[(((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048))];
    kernel_pad_shared[((((int)threadIdx.x) + 384))] = kernel[(((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 3072))];
    kernel_pad_shared[((((int)threadIdx.x) + 512))] = kernel[(((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096))];
    kernel_pad_shared[((((int)threadIdx.x) + 640))] = kernel[(((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 5120))];
    kernel_pad_shared[((((int)threadIdx.x) + 768))] = kernel[(((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144))];
    kernel_pad_shared[((((int)threadIdx.x) + 896))] = kernel[(((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168))];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[(0)] = data_pad_shared[(((ra_fused0_inner_outer * 64) + (((int)threadIdx.x) & 31)))];
      data_pad_shared_local[(1)] = data_pad_shared[((((ra_fused0_inner_outer * 64) + (((int)threadIdx.x) & 31)) + 32))];
      kernel_pad_shared_local[(0)] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 32) + ra_fused0_inner_outer))];
      kernel_pad_shared_local[(1)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 5) * 32) + ra_fused0_inner_outer) + 128))];
      kernel_pad_shared_local[(2)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 5) * 32) + ra_fused0_inner_outer) + 256))];
      kernel_pad_shared_local[(3)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 5) * 32) + ra_fused0_inner_outer) + 384))];
      kernel_pad_shared_local[(4)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 5) * 32) + ra_fused0_inner_outer) + 512))];
      kernel_pad_shared_local[(5)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 5) * 32) + ra_fused0_inner_outer) + 640))];
      kernel_pad_shared_local[(6)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 5) * 32) + ra_fused0_inner_outer) + 768))];
      kernel_pad_shared_local[(7)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 5) * 32) + ra_fused0_inner_outer) + 896))];
      conv_local[(0)] = (conv_local[(0)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(0)]));
      conv_local[(2)] = (conv_local[(2)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(1)]));
      conv_local[(4)] = (conv_local[(4)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(2)]));
      conv_local[(6)] = (conv_local[(6)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(3)]));
      conv_local[(8)] = (conv_local[(8)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(4)]));
      conv_local[(10)] = (conv_local[(10)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(5)]));
      conv_local[(12)] = (conv_local[(12)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(6)]));
      conv_local[(14)] = (conv_local[(14)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(7)]));
      conv_local[(1)] = (conv_local[(1)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(0)]));
      conv_local[(3)] = (conv_local[(3)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(1)]));
      conv_local[(5)] = (conv_local[(5)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(2)]));
      conv_local[(7)] = (conv_local[(7)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(3)]));
      conv_local[(9)] = (conv_local[(9)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(4)]));
      conv_local[(11)] = (conv_local[(11)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(5)]));
      conv_local[(13)] = (conv_local[(13)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(6)]));
      conv_local[(15)] = (conv_local[(15)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(7)]));
    }
  }
  conv_unpad[((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)))] = max((conv_local[(0)] + bias[((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 12544))] = max((conv_local[(2)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 4))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 25088))] = max((conv_local[(4)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 8))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 37632))] = max((conv_local[(6)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 12))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 50176))] = max((conv_local[(8)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 16))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 62720))] = max((conv_local[(10)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 20))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 75264))] = max((conv_local[(12)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 24))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 87808))] = max((conv_local[(14)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 28))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 32))] = max((conv_local[(1)] + bias[((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 12576))] = max((conv_local[(3)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 4))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 25120))] = max((conv_local[(5)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 8))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 37664))] = max((conv_local[(7)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 12))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 50208))] = max((conv_local[(9)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 16))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 62752))] = max((conv_local[(11)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 20))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 75296))] = max((conv_local[(13)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 24))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) % 49) * 64)) + (((int)threadIdx.x) & 31)) + 87840))] = max((conv_local[(15)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 5)) + 28))]), 0.000000e+00f);
}

dim3 grid(98, 1, 1);
dim3 block(128, 1, 1);
best_idx 10