#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[2];
  __shared__ float data_pad_shared[512];
  __shared__ float kernel_pad_shared[1024];
  float data_pad_shared_local[1];
  float kernel_pad_shared_local[2];
  conv_local[(0)] = 0.000000e+00f;
  conv_local[(1)] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 36; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[(((int)threadIdx.x))] = (((((0 < (((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) / 28) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 9) / 3))) && ((((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) / 28) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 9) / 3)) < 29)) && (0 < (((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) % 28) + (((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 3)))) && ((((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) % 28) + (((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 3)) < 29)) ? data[((((((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) / 9) * 784) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 9) / 3) * 28)) + ((((int)blockIdx.x) % 49) * 16)) + (((int)threadIdx.x) & 15)) + (((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 3)) - 29))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 256))] = (((((0 < (((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) / 28) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 7) % 9) / 3))) && ((((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) / 28) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 7) % 9) / 3)) < 29)) && (0 < (((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) % 28) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 1) % 3)))) && ((((((((int)blockIdx.x) % 49) * 16) + (((int)threadIdx.x) & 15)) % 28) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 1) % 3)) < 29)) ? data[(((((((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 16) / 9) * 784) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 7) % 9) / 3) * 28)) + ((((int)blockIdx.x) % 49) * 16)) + (((int)threadIdx.x) & 15)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 1) % 3)) - 29))] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 49) * 36864) + ((((int)threadIdx.x) >> 5) * 1152)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)))];
    kernel_pad_shared[((((int)threadIdx.x) + 256))] = kernel[(((((((((int)blockIdx.x) / 49) * 36864) + ((((int)threadIdx.x) >> 5) * 1152)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 9216))];
    kernel_pad_shared[((((int)threadIdx.x) + 512))] = kernel[(((((((((int)blockIdx.x) / 49) * 36864) + ((((int)threadIdx.x) >> 5) * 1152)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 18432))];
    kernel_pad_shared[((((int)threadIdx.x) + 768))] = kernel[(((((((((int)blockIdx.x) / 49) * 36864) + ((((int)threadIdx.x) >> 5) * 1152)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 27648))];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[(0)] = data_pad_shared[(((ra_fused0_inner_outer * 16) + (((int)threadIdx.x) & 15)))];
      kernel_pad_shared_local[(0)] = kernel_pad_shared[((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer))];
      kernel_pad_shared_local[(1)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer) + 512))];
      conv_local[(0)] = (conv_local[(0)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(0)]));
      conv_local[(1)] = (conv_local[(1)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(1)]));
    }
  }
  conv_unpad[((((((((int)blockIdx.x) / 49) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 49) * 16)) + (((int)threadIdx.x) & 15)))] = max((conv_local[(0)] + bias[((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 4)))]), 0.000000e+00f);
  conv_unpad[(((((((((int)blockIdx.x) / 49) * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 49) * 16)) + (((int)threadIdx.x) & 15)) + 12544))] = max((conv_local[(1)] + bias[(((((((int)blockIdx.x) / 49) * 32) + (((int)threadIdx.x) >> 4)) + 16))]), 0.000000e+00f);
}

dim3 grid(196, 1, 1);
dim3 block(256, 1, 1);
best_idx 5