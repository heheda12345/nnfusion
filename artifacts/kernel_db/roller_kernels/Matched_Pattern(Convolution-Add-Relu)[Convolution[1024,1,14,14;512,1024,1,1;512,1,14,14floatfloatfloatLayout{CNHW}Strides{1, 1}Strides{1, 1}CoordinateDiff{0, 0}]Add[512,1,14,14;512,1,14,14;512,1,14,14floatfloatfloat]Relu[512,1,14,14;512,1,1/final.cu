#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[2];
  __shared__ float data_pad_shared[512];
  __shared__ float kernel_pad_shared[1024];
  float data_pad_shared_local[1];
  float kernel_pad_shared_local[2];
  conv_local[(0)] = 0.000000e+00f;
  conv_local[(1)] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 32; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[(((int)threadIdx.x))] = (((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) < 196) ? data[(((((ra_fused0_outer * 6272) + ((((int)threadIdx.x) >> 4) * 196)) + ((((int)blockIdx.x) % 13) * 16)) + (((int)threadIdx.x) & 15)))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 256))] = (((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) < 196) ? data[((((((ra_fused0_outer * 6272) + ((((int)threadIdx.x) >> 4) * 196)) + ((((int)blockIdx.x) % 13) * 16)) + (((int)threadIdx.x) & 15)) + 3136))] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 13) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)))];
    kernel_pad_shared[((((int)threadIdx.x) + 256))] = kernel[(((((((((int)blockIdx.x) / 13) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192))];
    kernel_pad_shared[((((int)threadIdx.x) + 512))] = kernel[(((((((((int)blockIdx.x) / 13) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 16384))];
    kernel_pad_shared[((((int)threadIdx.x) + 768))] = kernel[(((((((((int)blockIdx.x) / 13) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 24576))];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[(0)] = data_pad_shared[(((ra_fused0_inner_outer * 16) + (((int)threadIdx.x) & 15)))];
      kernel_pad_shared_local[(0)] = kernel_pad_shared[((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer))];
      kernel_pad_shared_local[(1)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer) + 512))];
      conv_local[(0)] = (conv_local[(0)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(0)]));
      conv_local[(1)] = (conv_local[(1)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(1)]));
    }
  }
  if ((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) < 196) {
    conv_unpad[((((((((int)blockIdx.x) / 13) * 6272) + ((((int)threadIdx.x) >> 4) * 196)) + ((((int)blockIdx.x) % 13) * 16)) + (((int)threadIdx.x) & 15)))] = max((conv_local[(0)] + bias[((((((int)blockIdx.x) / 13) * 32) + (((int)threadIdx.x) >> 4)))]), 0.000000e+00f);
    conv_unpad[(((((((((int)blockIdx.x) / 13) * 6272) + ((((int)threadIdx.x) >> 4) * 196)) + ((((int)blockIdx.x) % 13) * 16)) + (((int)threadIdx.x) & 15)) + 3136))] = max((conv_local[(1)] + bias[(((((((int)blockIdx.x) / 13) * 32) + (((int)threadIdx.x) >> 4)) + 16))]), 0.000000e+00f);
  }
}

dim3 grid(208, 1, 1);
dim3 block(256, 1, 1);
best_idx 15