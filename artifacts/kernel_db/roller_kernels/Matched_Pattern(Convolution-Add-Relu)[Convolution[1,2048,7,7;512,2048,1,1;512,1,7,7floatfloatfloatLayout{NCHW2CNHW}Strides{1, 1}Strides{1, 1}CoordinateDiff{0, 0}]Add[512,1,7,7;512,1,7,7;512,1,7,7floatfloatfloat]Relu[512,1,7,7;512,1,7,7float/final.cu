#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[2];
  __shared__ float data_pad_shared[256];
  __shared__ float kernel_pad_shared[512];
  float data_pad_shared_local[1];
  float kernel_pad_shared_local[2];
  conv_local[(0)] = 0.000000e+00f;
  conv_local[(1)] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 64; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[(((int)threadIdx.x))] = (((((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) & 7)) < 49) ? data[(((((ra_fused0_outer * 1568) + ((((int)threadIdx.x) >> 3) * 49)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 64))] = (((((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) & 7)) < 49) ? data[((((((ra_fused0_outer * 1568) + ((((int)threadIdx.x) >> 3) * 49)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 392))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 128))] = (((((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) & 7)) < 49) ? data[((((((ra_fused0_outer * 1568) + ((((int)threadIdx.x) >> 3) * 49)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 784))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 192))] = (((((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) & 7)) < 49) ? data[((((((ra_fused0_outer * 1568) + ((((int)threadIdx.x) >> 3) * 49)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 1176))] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)))];
    kernel_pad_shared[((((int)threadIdx.x) + 64))] = kernel[(((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096))];
    kernel_pad_shared[((((int)threadIdx.x) + 128))] = kernel[(((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192))];
    kernel_pad_shared[((((int)threadIdx.x) + 192))] = kernel[(((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288))];
    kernel_pad_shared[((((int)threadIdx.x) + 256))] = kernel[(((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 16384))];
    kernel_pad_shared[((((int)threadIdx.x) + 320))] = kernel[(((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 20480))];
    kernel_pad_shared[((((int)threadIdx.x) + 384))] = kernel[(((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 24576))];
    kernel_pad_shared[((((int)threadIdx.x) + 448))] = kernel[(((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672))];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[(0)] = data_pad_shared[(((ra_fused0_inner_outer * 8) + (((int)threadIdx.x) & 7)))];
      kernel_pad_shared_local[(0)] = kernel_pad_shared[((((((int)threadIdx.x) >> 3) * 32) + ra_fused0_inner_outer))];
      kernel_pad_shared_local[(1)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 3) * 32) + ra_fused0_inner_outer) + 256))];
      conv_local[(0)] = (conv_local[(0)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(0)]));
      conv_local[(1)] = (conv_local[(1)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(1)]));
    }
  }
  if ((((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) & 7)) < 49) {
    conv_unpad[((((((((int)blockIdx.x) / 7) * 784) + ((((int)threadIdx.x) >> 3) * 49)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)))] = max((conv_local[(0)] + bias[((((((int)blockIdx.x) / 7) * 16) + (((int)threadIdx.x) >> 3)))]), 0.000000e+00f);
    conv_unpad[(((((((((int)blockIdx.x) / 7) * 784) + ((((int)threadIdx.x) >> 3) * 49)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 392))] = max((conv_local[(1)] + bias[(((((((int)blockIdx.x) / 7) * 16) + (((int)threadIdx.x) >> 3)) + 8))]), 0.000000e+00f);
  }
}

dim3 grid(224, 1, 1);
dim3 block(64, 1, 1);
best_idx 14