#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel0(float* __restrict__ data, float* __restrict__ avgpool2d) {
  float avgpool2d_local[2];
  __shared__ float padded_data_shared[1024];
  float padded_data_shared_local[2];
  avgpool2d_local[(0)] = 0.000000e+00f;
  avgpool2d_local[(1)] = 0.000000e+00f;
  padded_data_shared[(((int)threadIdx.x))] = data[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))];
  padded_data_shared[((((int)threadIdx.x) + 128))] = data[((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) + 128))];
  padded_data_shared[((((int)threadIdx.x) + 256))] = data[((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) + 256))];
  padded_data_shared[((((int)threadIdx.x) + 384))] = data[((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) + 384))];
  padded_data_shared[((((int)threadIdx.x) + 512))] = data[((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) + 512))];
  padded_data_shared[((((int)threadIdx.x) + 640))] = data[((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) + 640))];
  padded_data_shared[((((int)threadIdx.x) + 768))] = data[((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) + 768))];
  padded_data_shared[((((int)threadIdx.x) + 896))] = data[((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) + 896))];
  __syncthreads();
  for (int kh_inner_outer = 0; kh_inner_outer < 2; ++kh_inner_outer) {
    for (int kw_inner_outer = 0; kw_inner_outer < 2; ++kw_inner_outer) {
      padded_data_shared_local[(0)] = padded_data_shared[((((((((int)threadIdx.x) >> 2) * 32) + (kh_inner_outer * 16)) + ((((int)threadIdx.x) & 3) * 2)) + kw_inner_outer))];
      padded_data_shared_local[(1)] = padded_data_shared[(((((((((int)threadIdx.x) >> 2) * 32) + (kh_inner_outer * 16)) + ((((int)threadIdx.x) & 3) * 2)) + kw_inner_outer) + 8))];
      avgpool2d_local[(0)] = (avgpool2d_local[(0)] + (padded_data_shared_local[(0)] * 2.500000e-01f));
      avgpool2d_local[(1)] = (avgpool2d_local[(1)] + (padded_data_shared_local[(1)] * 2.500000e-01f));
    }
  }
  avgpool2d[((((((int)blockIdx.x) * 256) + ((((int)threadIdx.x) >> 2) * 8)) + (((int)threadIdx.x) & 3)))] = avgpool2d_local[(0)];
  avgpool2d[(((((((int)blockIdx.x) * 256) + ((((int)threadIdx.x) >> 2) * 8)) + (((int)threadIdx.x) & 3)) + 4))] = avgpool2d_local[(1)];
}

dim3 grid(512, 1, 1);
dim3 block(128, 1, 1);
best_idx 2