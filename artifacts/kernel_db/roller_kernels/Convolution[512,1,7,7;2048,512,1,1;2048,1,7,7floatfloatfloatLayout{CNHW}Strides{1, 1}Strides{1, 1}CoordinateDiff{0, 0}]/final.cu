#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv_unpad) {
  float conv_local[4];
  __shared__ float data_pad_shared[512];
  __shared__ float kernel_pad_shared[1024];
  float data_pad_shared_local[1];
  float kernel_pad_shared_local[4];
  conv_local[(0)] = 0.000000e+00f;
  conv_local[(1)] = 0.000000e+00f;
  conv_local[(2)] = 0.000000e+00f;
  conv_local[(3)] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 16; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[(((int)threadIdx.x))] = (((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 15)) < 49) ? data[(((((ra_fused0_outer * 1568) + ((((int)threadIdx.x) >> 4) * 49)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 128))] = (((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 15)) < 49) ? data[((((((ra_fused0_outer * 1568) + ((((int)threadIdx.x) >> 4) * 49)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)) + 392))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 256))] = (((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 15)) < 49) ? data[((((((ra_fused0_outer * 1568) + ((((int)threadIdx.x) >> 4) * 49)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)) + 784))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 384))] = (((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 15)) < 49) ? data[((((((ra_fused0_outer * 1568) + ((((int)threadIdx.x) >> 4) * 49)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)) + 1176))] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)))];
    kernel_pad_shared[((((int)threadIdx.x) + 128))] = kernel[(((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048))];
    kernel_pad_shared[((((int)threadIdx.x) + 256))] = kernel[(((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096))];
    kernel_pad_shared[((((int)threadIdx.x) + 384))] = kernel[(((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144))];
    kernel_pad_shared[((((int)threadIdx.x) + 512))] = kernel[(((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192))];
    kernel_pad_shared[((((int)threadIdx.x) + 640))] = kernel[(((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 10240))];
    kernel_pad_shared[((((int)threadIdx.x) + 768))] = kernel[(((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288))];
    kernel_pad_shared[((((int)threadIdx.x) + 896))] = kernel[(((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336))];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[(0)] = data_pad_shared[(((ra_fused0_inner_outer * 16) + (((int)threadIdx.x) & 15)))];
      kernel_pad_shared_local[(0)] = kernel_pad_shared[((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer))];
      kernel_pad_shared_local[(1)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer) + 256))];
      kernel_pad_shared_local[(2)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer) + 512))];
      kernel_pad_shared_local[(3)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer) + 768))];
      conv_local[(0)] = (conv_local[(0)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(0)]));
      conv_local[(1)] = (conv_local[(1)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(1)]));
      conv_local[(2)] = (conv_local[(2)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(2)]));
      conv_local[(3)] = (conv_local[(3)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(3)]));
    }
  }
  if ((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 15)) < 49) {
    conv_unpad[((((((((int)blockIdx.x) >> 2) * 1568) + ((((int)threadIdx.x) >> 4) * 49)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)))] = conv_local[(0)];
    conv_unpad[(((((((((int)blockIdx.x) >> 2) * 1568) + ((((int)threadIdx.x) >> 4) * 49)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)) + 392))] = conv_local[(1)];
    conv_unpad[(((((((((int)blockIdx.x) >> 2) * 1568) + ((((int)threadIdx.x) >> 4) * 49)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)) + 784))] = conv_local[(2)];
    conv_unpad[(((((((((int)blockIdx.x) >> 2) * 1568) + ((((int)threadIdx.x) >> 4) * 49)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)) + 1176))] = conv_local[(3)];
  }
}

dim3 grid(256, 1, 1);
dim3 block(128, 1, 1);
best_idx 15