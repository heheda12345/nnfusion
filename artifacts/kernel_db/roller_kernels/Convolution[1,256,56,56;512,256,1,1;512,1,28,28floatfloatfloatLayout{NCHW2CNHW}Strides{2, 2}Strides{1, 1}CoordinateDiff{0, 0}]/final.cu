#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(384) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv_unpad) {
  float conv_local[8];
  __shared__ float data_pad_shared[1024];
  __shared__ float kernel_pad_shared[3072];
  float data_pad_shared_local[2];
  float kernel_pad_shared_local[4];
  conv_local[(0)] = 0.000000e+00f;
  conv_local[(2)] = 0.000000e+00f;
  conv_local[(4)] = 0.000000e+00f;
  conv_local[(6)] = 0.000000e+00f;
  conv_local[(1)] = 0.000000e+00f;
  conv_local[(3)] = 0.000000e+00f;
  conv_local[(5)] = 0.000000e+00f;
  conv_local[(7)] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 8; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[(((int)threadIdx.x))] = (((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 31)) < 784) ? data[(((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 31)) / 28) * 112)) + (((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 31)) % 28) * 2)))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 384))] = (((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 31)) < 784) ? data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 31)) / 28) * 112)) + (((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 31)) % 28) * 2)) + 37632))] : 0.000000e+00f);
    if (((int)threadIdx.x) < 256) {
      data_pad_shared[((((int)threadIdx.x) + 768))] = (((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 31)) < 784) ? data[((((((ra_fused0_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + (((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 31)) / 28) * 112)) + (((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 31)) % 28) * 2)) + 75264))] : 0.000000e+00f);
    }
    kernel_pad_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 25) * 24576) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)))];
    kernel_pad_shared[((((int)threadIdx.x) + 384))] = kernel[(((((((((int)blockIdx.x) / 25) * 24576) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 3072))];
    kernel_pad_shared[((((int)threadIdx.x) + 768))] = (((((((int)blockIdx.x) / 25) * 96) + (((int)threadIdx.x) >> 5)) < 488) ? kernel[(((((((((int)blockIdx.x) / 25) * 24576) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144))] : 0.000000e+00f);
    kernel_pad_shared[((((int)threadIdx.x) + 1152))] = (((((((int)blockIdx.x) / 25) * 96) + (((int)threadIdx.x) >> 5)) < 476) ? kernel[(((((((((int)blockIdx.x) / 25) * 24576) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 9216))] : 0.000000e+00f);
    kernel_pad_shared[((((int)threadIdx.x) + 1536))] = (((((((int)blockIdx.x) / 25) * 96) + (((int)threadIdx.x) >> 5)) < 464) ? kernel[(((((((((int)blockIdx.x) / 25) * 24576) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288))] : 0.000000e+00f);
    kernel_pad_shared[((((int)threadIdx.x) + 1920))] = (((((((int)blockIdx.x) / 25) * 96) + (((int)threadIdx.x) >> 5)) < 452) ? kernel[(((((((((int)blockIdx.x) / 25) * 24576) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 15360))] : 0.000000e+00f);
    kernel_pad_shared[((((int)threadIdx.x) + 2304))] = (((((((int)blockIdx.x) / 25) * 96) + (((int)threadIdx.x) >> 5)) < 440) ? kernel[(((((((((int)blockIdx.x) / 25) * 24576) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 18432))] : 0.000000e+00f);
    kernel_pad_shared[((((int)threadIdx.x) + 2688))] = (((((((int)blockIdx.x) / 25) * 96) + (((int)threadIdx.x) >> 5)) < 428) ? kernel[(((((((((int)blockIdx.x) / 25) * 24576) + ((((int)threadIdx.x) >> 5) * 256)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 21504))] : 0.000000e+00f);
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[(0)] = data_pad_shared[(((ra_fused0_inner_outer * 32) + (((int)threadIdx.x) & 15)))];
      data_pad_shared_local[(1)] = data_pad_shared[((((ra_fused0_inner_outer * 32) + (((int)threadIdx.x) & 15)) + 16))];
      kernel_pad_shared_local[(0)] = kernel_pad_shared[((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer))];
      kernel_pad_shared_local[(1)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer) + 768))];
      kernel_pad_shared_local[(2)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer) + 1536))];
      kernel_pad_shared_local[(3)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer) + 2304))];
      conv_local[(0)] = (conv_local[(0)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(0)]));
      conv_local[(2)] = (conv_local[(2)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(1)]));
      conv_local[(4)] = (conv_local[(4)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(2)]));
      conv_local[(6)] = (conv_local[(6)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(3)]));
      conv_local[(1)] = (conv_local[(1)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(0)]));
      conv_local[(3)] = (conv_local[(3)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(1)]));
      conv_local[(5)] = (conv_local[(5)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(2)]));
      conv_local[(7)] = (conv_local[(7)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(3)]));
    }
  }
  conv_unpad[((((((((int)blockIdx.x) / 25) * 75264) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 25) * 32)) + (((int)threadIdx.x) & 15)))] = conv_local[(0)];
  if ((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 15)) < 768) {
    conv_unpad[(((((((((int)blockIdx.x) / 25) * 75264) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 25) * 32)) + (((int)threadIdx.x) & 15)) + 16))] = conv_local[(1)];
  }
  if ((((((int)blockIdx.x) / 25) * 96) + (((int)threadIdx.x) >> 4)) < 488) {
    conv_unpad[(((((((((int)blockIdx.x) / 25) * 75264) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 25) * 32)) + (((int)threadIdx.x) & 15)) + 18816))] = conv_local[(2)];
    if ((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 15)) < 768) {
      conv_unpad[(((((((((int)blockIdx.x) / 25) * 75264) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 25) * 32)) + (((int)threadIdx.x) & 15)) + 18832))] = conv_local[(3)];
    }
  }
  if ((((((int)blockIdx.x) / 25) * 96) + (((int)threadIdx.x) >> 4)) < 464) {
    conv_unpad[(((((((((int)blockIdx.x) / 25) * 75264) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 25) * 32)) + (((int)threadIdx.x) & 15)) + 37632))] = conv_local[(4)];
    if ((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 15)) < 768) {
      conv_unpad[(((((((((int)blockIdx.x) / 25) * 75264) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 25) * 32)) + (((int)threadIdx.x) & 15)) + 37648))] = conv_local[(5)];
    }
  }
  if ((((((int)blockIdx.x) / 25) * 96) + (((int)threadIdx.x) >> 4)) < 440) {
    conv_unpad[(((((((((int)blockIdx.x) / 25) * 75264) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 25) * 32)) + (((int)threadIdx.x) & 15)) + 56448))] = conv_local[(6)];
    if ((((((int)blockIdx.x) % 25) * 32) + (((int)threadIdx.x) & 15)) < 768) {
      conv_unpad[(((((((((int)blockIdx.x) / 25) * 75264) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) % 25) * 32)) + (((int)threadIdx.x) & 15)) + 56464))] = conv_local[(7)];
    }
  }
}

dim3 grid(150, 1, 1);
dim3 block(384, 1, 1);
best_idx 8