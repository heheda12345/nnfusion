#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[2];
  __shared__ float data_pad_shared[512];
  __shared__ float kernel_pad_shared[1024];
  float data_pad_shared_local[1];
  float kernel_pad_shared_local[2];
  conv_local[(0)] = 0.000000e+00f;
  conv_local[(1)] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 72; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[(((int)threadIdx.x))] = (((((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) < 196) && (0 < (((((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) % 196) / 14) * 2) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 9) / 3)))) && (0 < ((((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) % 14) * 2) + (((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 3)))) ? data[((((((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) / 9) * 784) + (((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) / 14) * 56)) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 9) / 3) * 28)) + (((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) % 14) * 2)) + (((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) % 3)) - 29))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 256))] = (((((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) < 196) && (0 < (((((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) % 196) / 14) * 2) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 7) % 9) / 3)))) && (0 < ((((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) % 14) * 2) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 1) % 3)))) ? data[(((((((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 16) / 9) * 784) + (((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) / 14) * 56)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 7) % 9) / 3) * 28)) + (((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) % 14) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 4)) + 1) % 3)) - 29))] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 13) * 73728) + ((((int)threadIdx.x) >> 5) * 2304)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)))];
    kernel_pad_shared[((((int)threadIdx.x) + 256))] = kernel[(((((((((int)blockIdx.x) / 13) * 73728) + ((((int)threadIdx.x) >> 5) * 2304)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 18432))];
    kernel_pad_shared[((((int)threadIdx.x) + 512))] = kernel[(((((((((int)blockIdx.x) / 13) * 73728) + ((((int)threadIdx.x) >> 5) * 2304)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 36864))];
    kernel_pad_shared[((((int)threadIdx.x) + 768))] = kernel[(((((((((int)blockIdx.x) / 13) * 73728) + ((((int)threadIdx.x) >> 5) * 2304)) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 55296))];
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[(0)] = data_pad_shared[(((ra_fused0_inner_outer * 16) + (((int)threadIdx.x) & 15)))];
      kernel_pad_shared_local[(0)] = kernel_pad_shared[((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer))];
      kernel_pad_shared_local[(1)] = kernel_pad_shared[(((((((int)threadIdx.x) >> 4) * 32) + ra_fused0_inner_outer) + 512))];
      conv_local[(0)] = (conv_local[(0)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(0)]));
      conv_local[(1)] = (conv_local[(1)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(1)]));
    }
  }
  if ((((((int)blockIdx.x) % 13) * 16) + (((int)threadIdx.x) & 15)) < 196) {
    conv_unpad[((((((((int)blockIdx.x) / 13) * 6272) + ((((int)threadIdx.x) >> 4) * 196)) + ((((int)blockIdx.x) % 13) * 16)) + (((int)threadIdx.x) & 15)))] = max((conv_local[(0)] + bias[((((((int)blockIdx.x) / 13) * 32) + (((int)threadIdx.x) >> 4)))]), 0.000000e+00f);
    conv_unpad[(((((((((int)blockIdx.x) / 13) * 6272) + ((((int)threadIdx.x) >> 4) * 196)) + ((((int)blockIdx.x) % 13) * 16)) + (((int)threadIdx.x) & 15)) + 3136))] = max((conv_local[(1)] + bias[(((((((int)blockIdx.x) / 13) * 32) + (((int)threadIdx.x) >> 4)) + 16))]), 0.000000e+00f);
  }
}

dim3 grid(104, 1, 1);
dim3 block(256, 1, 1);
best_idx 7